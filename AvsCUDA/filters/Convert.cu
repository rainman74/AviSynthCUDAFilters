#include "hip/hip_runtime.h"
// Avisynth v2.5.  Copyright 2002-2009 Ben Rudiak-Gould et al.
// http://www.avisynth.org

// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 2 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA, or visit
// http://www.gnu.org/copyleft/gpl.html .
//
// Linking Avisynth statically or dynamically with other modules is making a
// combined work based on Avisynth.  Thus, the terms and conditions of the GNU
// General Public License cover the whole combination.
//
// As a special exception, the copyright holders of Avisynth give you
// permission to link Avisynth with independent modules that communicate with
// Avisynth solely through the interfaces defined in avisynth.h, regardless of the license
// terms of these independent modules, and to copy and distribute the
// resulting combined work under terms of your choice, provided that
// every copy of the combined work is accompanied by a complete copy of
// the source code of Avisynth (the version of Avisynth used to produce the
// combined work), being distributed under the terms of the GNU General
// Public License plus this exception.  An independent module is a module
// which is not derived from or based on Avisynth, such as 3rd-party filters,
// import and export plugins, or graphical user interfaces.


#include "../AvsCUDA.h"
#include <avs/alignment.h>
#include "rgy_osdep.h"
#include <smmintrin.h>
#include <tuple>
#include <map>

#include "convert_avx.h"
#include "convert_avx2.h"

#include "Copy.h"


RGY_TARGET("sse4.1")
static RGY_FORCEINLINE __m128i af_mm_min_epu16_sse41(__m128i a, __m128i b) {
	return _mm_min_epu16(a, b);
}

template<bool useSSE41>
static RGY_FORCEINLINE __m128i af_mm_min_epu16(__m128i a, __m128i b) {
	if constexpr(useSSE41) {
		return af_mm_min_epu16_sse41(a, b);
	} else {
		return _MM_MIN_EPU16(a, b);
	}
}

RGY_TARGET("sse4.1")
static RGY_FORCEINLINE __m128i af_mm_packus_epi32_sse41(__m128i a, __m128i b) {
	return _mm_packus_epi32(a, b);
}

template<bool useSSE41>
static RGY_FORCEINLINE __m128i af_mm_packus_epi32(__m128i a, __m128i b) {
	if constexpr(useSSE41) {
		return af_mm_packus_epi32_sse41(a, b);
	} else {
		return _MM_PACKUS_EPI32(a, b);
	}
}

//--------------- planar bit depth conversions
// todo: separate file?
typedef void(*BitDepthConvFuncPtr)(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch);
typedef void(*BitDepthConvFuncPtr2)(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env);

class ConvertBits : public GenericVideoFilter
{
public:
	ConvertBits(PClip _child, const int _dither_mode, const int _target_bitdepth, bool _truerange, bool _fulls, bool _fulld, int _dither_bitdepth, IScriptEnvironment* env);
	PVideoFrame __stdcall GetFrame(int n, IScriptEnvironment* env);

	int __stdcall SetCacheHints(int cachehints, int frame_range) override {
		if (cachehints == CACHE_GET_DEV_TYPE) {
			return GetDeviceTypes(child) &
				(DEV_TYPE_CPU | DEV_TYPE_CUDA);
		}
		return cachehints == CACHE_GET_MTMODE ? MT_NICE_FILTER : 0;
	}

	static AVSValue __cdecl Create(AVSValue args, void*, IScriptEnvironment* env);
private:
	BitDepthConvFuncPtr conv_function;
	BitDepthConvFuncPtr conv_function_chroma; // 32bit float YUV chroma
	BitDepthConvFuncPtr conv_function_a;
	BitDepthConvFuncPtr2 conv_cuda;
	BitDepthConvFuncPtr2 conv_cuda_chroma;
	float float_range;
	int dither_mode;
	int pixelsize;
	int bits_per_pixel;
	int target_bitdepth;
	int dither_bitdepth;
	bool truerange; // if 16->10 range reducing or e.g. 14->16 bit range expansion needed
	bool format_change_only;
	bool fulls; // source is full range (defaults: rgb=true, yuv=false (bit shift))
	bool fulld; // destination is full range (defaults: rgb=true, yuv=false (bit shift))
};

/********************************************************************
***** Declare index of new filters for Avisynth's filter engine *****
********************************************************************/

extern const FuncDefinition Convert_filters[] = {       // matrix can be "rec601", "rec709", "PC.601" or "PC.709" or "rec2020"
	{ "ConvertTo8bit",  BUILTIN_FUNC_PREFIX, "c[bits]i[truerange]b[dither]f[dither_bits]i[fulls]b[fulld]b", ConvertBits::Create, (void *)8 },
	{ "ConvertTo16bit", BUILTIN_FUNC_PREFIX, "c[bits]i[truerange]b[dither]f[dither_bits]i[fulls]b[fulld]b", ConvertBits::Create, (void *)16 },
	{ "ConvertToFloat", BUILTIN_FUNC_PREFIX, "c[bits]i[truerange]b[dither]f[dither_bits]i[fulls]b[fulld]b", ConvertBits::Create, (void *)32 },
	{ "ConvertBits",    BUILTIN_FUNC_PREFIX, "c[bits]i[truerange]b[dither]f[dither_bits]i[fulls]b[fulld]b", ConvertBits::Create, (void *)0 },
	{ 0 }
};

template <int SHIFT> struct Dither {
	static __device__ BYTE get(int x, int y) { return 0; }
};

__constant__ BYTE c_dither2[2][2] = {
	{ 0, 2 },
	{ 3, 1 }
};
template <> struct Dither<2> {
	static __device__ BYTE get(int x, int y) { return c_dither2[y][x]; }
};

__constant__ BYTE c_dither4[4][4] = {
	{ 0,  8,  2, 10 },
	{ 12,  4, 14,  6 },
	{ 3, 11,  1,  9 },
	{ 15,  7, 13,  5 }
};
template <> struct Dither<4> {
	static __device__ BYTE get(int x, int y) { return c_dither4[y][x]; }
};

__constant__ BYTE c_dither6[8][8] = {
	{ 0, 32,  8, 40,  2, 34, 10, 42 }, /* 8x8 Bayer ordered dithering */
	{ 48, 16, 56, 24, 50, 18, 58, 26 }, /* pattern. Each input pixel */
	{ 12, 44,  4, 36, 14, 46,  6, 38 }, /* is scaled to the 0..63 range */
	{ 60, 28, 52, 20, 62, 30, 54, 22 }, /* before looking in this table */
	{ 3, 35, 11, 43,  1, 33,  9, 41 }, /* to determine the action. */
	{ 51, 19, 59, 27, 49, 17, 57, 25 },
	{ 15, 47,  7, 39, 13, 45,  5, 37 },
	{ 63, 31, 55, 23, 61, 29, 53, 21 }
};
template <> struct Dither<6> {
	static __device__ BYTE get(int x, int y) { return c_dither6[y][x]; }
};

__constant__ BYTE c_dither8[16][16] = {
	{ 0,192, 48,240, 12,204, 60,252,  3,195, 51,243, 15,207, 63,255 },
	{ 128, 64,176,112,140, 76,188,124,131, 67,179,115,143, 79,191,127 },
	{ 32,224, 16,208, 44,236, 28,220, 35,227, 19,211, 47,239, 31,223 },
	{ 160, 96,144, 80,172,108,156, 92,163, 99,147, 83,175,111,159, 95 },
	{ 8,200, 56,248,  4,196, 52,244, 11,203, 59,251,  7,199, 55,247 },
	{ 136, 72,184,120,132, 68,180,116,139, 75,187,123,135, 71,183,119 },
	{ 40,232, 24,216, 36,228, 20,212, 43,235, 27,219, 39,231, 23,215 },
	{ 168,104,152, 88,164,100,148, 84,171,107,155, 91,167,103,151, 87 },
	{ 2,194, 50,242, 14,206, 62,254,  1,193, 49,241, 13,205, 61,253 },
	{ 130, 66,178,114,142, 78,190,126,129, 65,177,113,141, 77,189,125 },
	{ 34,226, 18,210, 46,238, 30,222, 33,225, 17,209, 45,237, 29,221 },
	{ 162, 98,146, 82,174,110,158, 94,161, 97,145, 81,173,109,157, 93 },
	{ 10,202, 58,250,  6,198, 54,246,  9,201, 57,249,  5,197, 53,245 },
	{ 138, 74,186,122,134, 70,182,118,137, 73,185,121,133, 69,181,117 },
	{ 42,234, 26,218, 38,230, 22,214, 41,233, 25,217, 37,229, 21,213 },
	{ 170,106,154, 90,166,102,150, 86,169,105,153, 89,165,101,149, 85 }
};
template <> struct Dither<8> {
	static __device__ BYTE get(int x, int y) { return c_dither8[y][x]; }
};

template <int TYPE>
struct ConvertBitsKernel { };

template <int BITS> struct BitsToType { typedef uint16_t type; };
template <> struct BitsToType<8> { typedef uint8_t type; };
template <> struct BitsToType<32> { typedef float type; };

// 0: copy
template <> struct ConvertBitsKernel<0> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		Copy(dstp, dst_pitch, srcp, src_pitch, src_rowsize, src_height, env);
	}
};

template <typename SRC_TYPE, typename TGT_TYPE, int SHIFT, int TGT_BITS>
__global__ void kl_convert_to_lower_bits_dither(int width, int height,
	TGT_TYPE* dst, int dst_pitch, const SRC_TYPE* src, int src_pitch)
{
	enum { 
		DITHER_W = (1 << (SHIFT >> 1)), 
		MASK = DITHER_W - 1
	};

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		auto tmp = (src[x + y * src_pitch] + Dither<SHIFT>::get(x & MASK, y & MASK)) >> SHIFT;
		dst[x + y * dst_pitch] = (TGT_TYPE)min(tmp, (1 << TGT_BITS) - 1);
	}
}

// 1: convert_to_lower_bits_dither
template <> struct ConvertBitsKernel<1> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		typedef typename BitsToType<SRC_BITS>::type SRC_TYPE;
		typedef typename BitsToType<TGT_BITS>::type TGT_TYPE;
		int width = src_rowsize / sizeof(SRC_TYPE);
		int height = src_height;
		dim3 threads(32, 16);
		dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
		kl_convert_to_lower_bits_dither
			<SRC_TYPE, TGT_TYPE, SRC_BITS - TGT_BITS, TGT_BITS>
			<< <blocks, threads >> > (
				width, height,
				(TGT_TYPE*)dstp, dst_pitch / sizeof(TGT_TYPE),
				(SRC_TYPE*)srcp, src_pitch / sizeof(SRC_TYPE));
		DEBUG_SYNC;
	}
};

template <typename SRC_TYPE, typename TGT_TYPE, int SHIFT, int TGT_BITS>
__global__ void kl_convert_to_lower_bits_no_dither(int width, int height,
	TGT_TYPE* dst, int dst_pitch, const SRC_TYPE* src, int src_pitch)
{
	enum { 
		HALF = (1 << (SHIFT - 1))
	};

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		//auto tmp = (src[x + y * src_pitch] + HALF) >> SHIFT;
		auto tmp = (src[x + y * src_pitch]) >> SHIFT;
		dst[x + y * dst_pitch] = (TGT_TYPE)min(tmp, (1 << TGT_BITS) - 1);
	}
}

// 2: convert_to_lower_bits_no_dither
template <> struct ConvertBitsKernel<2> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		typedef typename BitsToType<SRC_BITS>::type SRC_TYPE;
		typedef typename BitsToType<TGT_BITS>::type TGT_TYPE;
		int width = src_rowsize / sizeof(SRC_TYPE);
		int height = src_height;
		dim3 threads(32, 16);
		dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
		kl_convert_to_lower_bits_no_dither
			<SRC_TYPE, TGT_TYPE, SRC_BITS - TGT_BITS, TGT_BITS>
			<< <blocks, threads >> > (
				width, height,
				(TGT_TYPE*)dstp, dst_pitch / sizeof(TGT_TYPE),
				(SRC_TYPE*)srcp, src_pitch / sizeof(SRC_TYPE));
		DEBUG_SYNC;
	}
};

template <typename SRC_TYPE, typename TGT_TYPE, int SHIFT, int TGT_BITS>
__global__ void kl_convert_to_higher_bits(int width, int height, 
	TGT_TYPE* dst, int dst_pitch, const SRC_TYPE* src, int src_pitch)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height) {
		dst[x + y * dst_pitch] = (TGT_TYPE)min(src[x + y * src_pitch] << SHIFT, (1 << TGT_BITS) - 1);
	}
}

// 3: convert_to_higher_bits
template <> struct ConvertBitsKernel<3> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		typedef typename BitsToType<SRC_BITS>::type SRC_TYPE;
		typedef typename BitsToType<TGT_BITS>::type TGT_TYPE;
		int width = src_rowsize / sizeof(SRC_TYPE);
		int height = src_height;
		dim3 threads(32, 16);
		dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
		kl_convert_to_higher_bits
			<SRC_TYPE, TGT_TYPE, TGT_BITS - SRC_BITS, TGT_BITS>
			<< <blocks, threads >> > (
				width, height,
				(TGT_TYPE*)dstp, dst_pitch / sizeof(TGT_TYPE),
				(SRC_TYPE*)srcp, src_pitch / sizeof(SRC_TYPE));
		DEBUG_SYNC;
	}
};

template <typename TGT_TYPE, int TGT_BITS, bool CHROMA>
__global__ void kl_convert_from_float(int width, int height,
	TGT_TYPE* dst, int dst_pitch, const float* src, int src_pitch)
{
	constexpr float MAX_VAL = 255 << (TGT_BITS - 8);
	constexpr float HALF = 128 << (TGT_BITS - 8);
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height) {
		float tmp;
		if (CHROMA) {
			tmp = src[x + y * src_pitch] * MAX_VAL + HALF + 0.5f;
		}
		else {
			tmp = src[x + y * src_pitch] * MAX_VAL + 0.5f;
		}
		dst[x + y * dst_pitch] = (TGT_TYPE)clamp(tmp, 0.0f, (float)MAX_VAL);
	}
}

// 4: convert_from_float
template <> struct ConvertBitsKernel<4> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		typedef float SRC_TYPE;
		typedef typename BitsToType<TGT_BITS>::type TGT_TYPE;
		int width = src_rowsize / sizeof(SRC_TYPE);
		int height = src_height;
		dim3 threads(32, 16);
		dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
		kl_convert_from_float
			<TGT_TYPE, TGT_BITS, CHROMA>
			<< <blocks, threads >> > (
				width, height,
				(TGT_TYPE*)dstp, dst_pitch / sizeof(TGT_TYPE),
				(SRC_TYPE*)srcp, src_pitch / sizeof(SRC_TYPE));
		DEBUG_SYNC;
	}
};

template <typename SRC_TYPE, int SRC_BITS, bool CHROMA>
__global__ void kl_convert_to_float(int width, int height,
	float* dst, int dst_pitch, const SRC_TYPE* src, int src_pitch)
{
	constexpr float MAX_VAL = 255 << (SRC_BITS - 8);
	constexpr float FACTOR = 1.0f / MAX_VAL;
	constexpr float HALF = 128 << (SRC_BITS - 8);
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < width && y < height) {
		float tmp;
		if (CHROMA) {
			tmp = (float)(src[x + y * src_pitch] - HALF) * FACTOR;
		}
		else {
			tmp = (float)src[x + y * src_pitch] * FACTOR;
		}
		dst[x + y * dst_pitch] = tmp;
	}
}

// 5: convert_to_float
template <> struct ConvertBitsKernel<5> {
	template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
	static void launch(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch, PNeoEnv env)
	{
		typedef typename BitsToType<SRC_BITS>::type SRC_TYPE;
		typedef float TGT_TYPE;
		int width = src_rowsize / sizeof(SRC_TYPE);
		int height = src_height;
		dim3 threads(32, 16);
		dim3 blocks(nblocks(width, threads.x), nblocks(height, threads.y));
		kl_convert_to_float
			<SRC_TYPE, SRC_BITS, CHROMA>
			<< <blocks, threads >> > (
				width, height,
				(TGT_TYPE*)dstp, dst_pitch / sizeof(TGT_TYPE),
				(SRC_TYPE*)srcp, src_pitch / sizeof(SRC_TYPE));
		DEBUG_SYNC;
	}
};


/**********************************
******  Bitdepth conversions  *****
**********************************/
// 10->8
// repeated 4x for sse size 16
static const struct dither2x2_t
{
	const BYTE data[4] = {
		0, 2,
		3, 1,
	};
	// cycle: 2
	alignas(16) const BYTE data_sse2[2 * 16] = {
		0, 2, 0, 2, 0, 2, 0, 2, 0, 2, 0, 2, 0, 2, 0, 2,
		3, 1, 3, 1, 3, 1, 3, 1, 3, 1, 3, 1, 3, 1, 3, 1
	};
	dither2x2_t() {};
} dither2x2;


// 12->8
static const struct dither4x4_t
{
	const BYTE data[16] = {
		0,  8,  2, 10,
		12,  4, 14,  6,
		3, 11,  1,  9,
		15,  7, 13,  5
	};
	// cycle: 4
	alignas(16) const BYTE data_sse2[4 * 16] = {
		0,  8,  2, 10,  0,  8,  2, 10,  0,  8,  2, 10,  0,  8,  2, 10,
		12,  4, 14,  6, 12,  4, 14,  6, 12,  4, 14,  6, 12,  4, 14,  6,
		3, 11,  1,  9,  3, 11,  1,  9,  3, 11,  1,  9,  3, 11,  1,  9,
		15,  7, 13,  5, 15,  7, 13,  5, 15,  7, 13,  5, 15,  7, 13,  5
	};
	dither4x4_t() {};
} dither4x4;

// 14->8
static const struct dither8x8_t
{
	const BYTE data[8][8] = {
		{ 0, 32,  8, 40,  2, 34, 10, 42 }, /* 8x8 Bayer ordered dithering */
		{ 48, 16, 56, 24, 50, 18, 58, 26 }, /* pattern. Each input pixel */
		{ 12, 44,  4, 36, 14, 46,  6, 38 }, /* is scaled to the 0..63 range */
		{ 60, 28, 52, 20, 62, 30, 54, 22 }, /* before looking in this table */
		{ 3, 35, 11, 43,  1, 33,  9, 41 }, /* to determine the action. */
		{ 51, 19, 59, 27, 49, 17, 57, 25 },
		{ 15, 47,  7, 39, 13, 45,  5, 37 },
		{ 63, 31, 55, 23, 61, 29, 53, 21 }
	};
	// cycle: 8
	alignas(16) const BYTE data_sse2[8][16] = {
		{ 0, 32,  8, 40,  2, 34, 10, 42,  0, 32,  8, 40,  2, 34, 10, 42 }, /* 8x8 Bayer ordered dithering */
		{ 48, 16, 56, 24, 50, 18, 58, 26, 48, 16, 56, 24, 50, 18, 58, 26 }, /* pattern. Each input pixel */
		{ 12, 44,  4, 36, 14, 46,  6, 38, 12, 44,  4, 36, 14, 46,  6, 38 }, /* is scaled to the 0..63 range */
		{ 60, 28, 52, 20, 62, 30, 54, 22, 60, 28, 52, 20, 62, 30, 54, 22 }, /* before looking in this table */
		{ 3, 35, 11, 43,  1, 33,  9, 41,  3, 35, 11, 43,  1, 33,  9, 41 }, /* to determine the action. */
		{ 51, 19, 59, 27, 49, 17, 57, 25, 51, 19, 59, 27, 49, 17, 57, 25 },
		{ 15, 47,  7, 39, 13, 45,  5, 37, 15, 47,  7, 39, 13, 45,  5, 37 },
		{ 63, 31, 55, 23, 61, 29, 53, 21, 63, 31, 55, 23, 61, 29, 53, 21 }
	};
	dither8x8_t() {};
} dither8x8;

// 16->8
static const struct dither16x16_t
{
	// cycle: 16x
	alignas(16) const BYTE data[16][16] = {
		{ 0,192, 48,240, 12,204, 60,252,  3,195, 51,243, 15,207, 63,255 },
		{ 128, 64,176,112,140, 76,188,124,131, 67,179,115,143, 79,191,127 },
		{ 32,224, 16,208, 44,236, 28,220, 35,227, 19,211, 47,239, 31,223 },
		{ 160, 96,144, 80,172,108,156, 92,163, 99,147, 83,175,111,159, 95 },
		{ 8,200, 56,248,  4,196, 52,244, 11,203, 59,251,  7,199, 55,247 },
		{ 136, 72,184,120,132, 68,180,116,139, 75,187,123,135, 71,183,119 },
		{ 40,232, 24,216, 36,228, 20,212, 43,235, 27,219, 39,231, 23,215 },
		{ 168,104,152, 88,164,100,148, 84,171,107,155, 91,167,103,151, 87 },
		{ 2,194, 50,242, 14,206, 62,254,  1,193, 49,241, 13,205, 61,253 },
		{ 130, 66,178,114,142, 78,190,126,129, 65,177,113,141, 77,189,125 },
		{ 34,226, 18,210, 46,238, 30,222, 33,225, 17,209, 45,237, 29,221 },
		{ 162, 98,146, 82,174,110,158, 94,161, 97,145, 81,173,109,157, 93 },
		{ 10,202, 58,250,  6,198, 54,246,  9,201, 57,249,  5,197, 53,245 },
		{ 138, 74,186,122,134, 70,182,118,137, 73,185,121,133, 69,181,117 },
		{ 42,234, 26,218, 38,230, 22,214, 41,233, 25,217, 37,229, 21,213 },
		{ 170,106,154, 90,166,102,150, 86,169,105,153, 89,165,101,149, 85 }
	};
	dither16x16_t() {};
} dither16x16;

template<uint8_t sourcebits, int dither_mode, int TARGET_DITHER_BITDEPTH, int rgb_step>
static void convert_rgb_uint16_to_8_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp0 = reinterpret_cast<const uint16_t *>(srcp);
	src_pitch = src_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);

	int _y = 0; // for ordered dither

	const int TARGET_BITDEPTH = 8; // here is constant (uint8_t target)

																 // for test, make it 2,4,6,8. sourcebits-TARGET_DITHER_BITDEPTH cannot exceed 8 bit
																 // const int TARGET_DITHER_BITDEPTH = 2;

	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1;
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but generality)
	const int DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const int DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const int MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data); break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data); break;
	case 6: matrix = reinterpret_cast<const BYTE *>(dither8x8.data); break;
	case 8: matrix = reinterpret_cast<const BYTE *>(dither16x16.data); break;
  default: return; // n/a
	}

	for (int y = 0; y<src_height; y++)
	{
    if constexpr(dither_mode == 0)
			_y = (y & MASK) << DITHER_ORDER; // ordered dither
		for (int x = 0; x < src_width; x++)
		{
      if constexpr(dither_mode < 0) // -1: no dither
			{
				const float mulfactor = sourcebits == 16 ? (1.0f / 257.0f) :
					sourcebits == 14 ? (255.0f / 16383.0f) :
					sourcebits == 12 ? (255.0f / 4095.0f) :
					(255.0f / 1023.0f); // 10 bits

				dstp[x] = (uint8_t)(srcp0[x] * mulfactor + 0.5f);
				// C cast truncates, use +0.5f rounder, which uses cvttss2si

				// old method: no rounding but fast
				// no integer division (fast tricky algorithm by compiler), rounding problems, pic gets darker
				// dstp[x] = srcp0[x] / 257; // RGB: full range 0..255 <-> 0..65535 (*255 / 65535)
				// dstp[x] = srcp0[x] * 255 / 16383; // RGB: full range 0..255 <-> 0..16384-1
				// dstp[x] = srcp0[x] * 255 / 4095; // RGB: full range 0..255 <-> 0..4096-1
				// dstp[x] = srcp0[x] * 255 / 1023; // RGB: full range 0..255 <-> 0..1024-1
			}
			else { // dither_mode == 0 -> ordered dither
				const int corr = matrix[_y | ((x / rgb_step) & MASK)];
				// vvv for the non-fullscale version: int new_pixel = ((srcp0[x] + corr) >> DITHER_BIT_DIFF);
				int new_pixel;

				const float mulfactor =
					DITHER_BIT_DIFF == 8 ? (1.0f / 257.0f) :
					DITHER_BIT_DIFF == 6 ? (255.0f / 16383.0f) :
					DITHER_BIT_DIFF == 4 ? (255.0f / 4095.0f) :
					DITHER_BIT_DIFF == 2 ? (255.0f / 1023.0f) : // 10 bits
					1.0f;

        if constexpr(TARGET_DITHER_BITDEPTH <= 4)
					new_pixel = (uint16_t)((srcp0[x] + corr) * mulfactor); // rounding here makes brightness shift
        else if constexpr(DITHER_BIT_DIFF > 0)
					new_pixel = (uint16_t)((srcp0[x] + corr) * mulfactor + 0.5f);
				else
					new_pixel = (uint16_t)(srcp0[x] + corr);

				new_pixel = min(new_pixel, max_pixel_value_dithered); // clamp upper

																															// scale back to the required bit depth
																															// for generality. Now target == 8 bit, and dither_target is also 8 bit
																															// for test: source:10 bit, target=8 bit, dither_target=4 bit
				const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
        if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0)  // dither to 8, target to 8
					new_pixel = new_pixel << BITDIFF_BETWEEN_DITHER_AND_TARGET; // if implemented non-8bit dither target, this should be fullscale
				dstp[x] = (BYTE)new_pixel;
			}
		} // x
		dstp += dst_pitch;
		srcp0 += src_pitch;
	}
}

template<uint8_t sourcebits, int dither_mode, int TARGET_DITHER_BITDEPTH, int rgb_step>
static void convert_rgb_uint16_to_8_sse2(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp0 = reinterpret_cast<const uint16_t *>(srcp);
	src_pitch = src_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);

	int _y = 0; // for ordered dither

	const int TARGET_BITDEPTH = 8; // here is constant (uint8_t target)

																 // for test, make it 2,4,6,8. sourcebits-TARGET_DITHER_BITDEPTH cannot exceed 8 bit
																 // const int TARGET_DITHER_BITDEPTH = 2;

	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1;
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but generality)
	const int DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const int DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const int MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data); break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data); break;
	case 6: matrix = reinterpret_cast<const BYTE *>(dither8x8.data); break;
	case 8: matrix = reinterpret_cast<const BYTE *>(dither16x16.data); break;
  default: return; // n/a
	}

	// 20171024: given up integer division, rounding problems
	const float mulfactor =
		sourcebits == 16 ? (1.0f / 257.0f) :
		sourcebits == 14 ? (255.0f / 16383.0f) :
		sourcebits == 12 ? (255.0f / 4095.0f) :
		(255.0f / 1023.0f); // 10 bits
	const __m128 mulfactor_simd = _mm_set1_ps(mulfactor);
	const __m128i zero = _mm_setzero_si128();

	for (int y = 0; y < src_height; y++)
	{
    if constexpr(dither_mode == 0)
			_y = (y & MASK) << DITHER_ORDER; // ordered dither
		for (int x = 0; x < src_width; x += 8) // 8 * uint16_t at a time
		{
      if constexpr(dither_mode < 0) // -1: no dither
			{

				// C: dstp[x] = (uint8_t)(srcp0[x] * mulfactor + 0.5f);
				// C cast truncates, use +0.5f rounder, which uses cvttss2si

				__m128i pixel_i = _mm_load_si128(reinterpret_cast<const __m128i *>(srcp0 + x)); // 16 bytes 8 pixels

				__m128 pixel_f_lo = _mm_cvtepi32_ps(_mm_unpacklo_epi16(pixel_i, zero)); // 4 floats
				__m128 mulled_lo = _mm_mul_ps(pixel_f_lo, mulfactor_simd);
				__m128i converted32_lo = _mm_cvtps_epi32(mulled_lo); // rounding ok, nearest. no +0.5 needed

				__m128 pixel_f_hi = _mm_cvtepi32_ps(_mm_unpackhi_epi16(pixel_i, zero)); // 4 floats
				__m128 mulled_hi = _mm_mul_ps(pixel_f_hi, mulfactor_simd);
				__m128i converted32_hi = _mm_cvtps_epi32(mulled_hi);

				__m128i converted_16 = _mm_packs_epi32(converted32_lo, converted32_hi);
				__m128i converted_8 = _mm_packus_epi16(converted_16, zero);
				_mm_storel_epi64(reinterpret_cast<__m128i *>(&dstp[x]), converted_8); // store 8 bytes
			}
			else { // dither_mode == 0 -> ordered dither
						 //  const int corr = matrix[_y | ((x / rgb_step) & MASK)];
				__m128i corr_lo = _mm_set_epi32(
					matrix[_y | (((x + 3) / rgb_step) & MASK)],
					matrix[_y | (((x + 2) / rgb_step) & MASK)],
					matrix[_y | (((x + 1) / rgb_step) & MASK)],
					matrix[_y | (((x + 0) / rgb_step) & MASK)]
				);
				__m128i corr_hi = _mm_set_epi32(
					matrix[_y | (((x + 7) / rgb_step) & MASK)],
					matrix[_y | (((x + 6) / rgb_step) & MASK)],
					matrix[_y | (((x + 5) / rgb_step) & MASK)],
					matrix[_y | (((x + 4) / rgb_step) & MASK)]
				);
				// vvv for the non-fullscale version: int new_pixel = ((srcp0[x] + corr) >> DITHER_BIT_DIFF);

				// no integer division, rounding problems
				const float mulfactor_dith =
					DITHER_BIT_DIFF == 8 ? (1.0f / 257.0f) :
					DITHER_BIT_DIFF == 6 ? (255.0f / 16383.0f) :
					DITHER_BIT_DIFF == 4 ? (255.0f / 4095.0f) :
					DITHER_BIT_DIFF == 2 ? (255.0f / 1023.0f) :
					1.0f;
				__m128 mulfactor_dith_simd = _mm_set1_ps(mulfactor_dith);

				__m128i pixel_i = _mm_load_si128(reinterpret_cast<const __m128i *>(srcp0 + x)); // 16 bytes 8 pixels
				__m128i pixel_i_lo = _mm_add_epi32(_mm_unpacklo_epi16(pixel_i, zero), corr_lo);
				__m128i pixel_i_hi = _mm_add_epi32(_mm_unpackhi_epi16(pixel_i, zero), corr_hi);
				__m128i converted32_lo, converted32_hi;

				/* C:
				if (TARGET_DITHER_BITDEPTH <= 4)
				new_pixel = (uint16_t)((srcp0[x] + corr) * mulfactor); // rounding here makes brightness shift
				else if (DITHER_BIT_DIFF > 0)
				new_pixel = (uint16_t)((srcp0[x] + corr) * mulfactor + 0.5f);
				else
				new_pixel = (uint16_t)(srcp0[x] + corr);
				*/
        if constexpr(TARGET_DITHER_BITDEPTH <= 4) {
					// round: truncate
					__m128 pixel_f_lo = _mm_cvtepi32_ps(pixel_i_lo); // 4 floats
					__m128 mulled_lo = _mm_mul_ps(pixel_f_lo, mulfactor_dith_simd);
					converted32_lo = _mm_cvttps_epi32(mulled_lo); // truncate! rounding here makes brightness shift

					__m128 pixel_f_hi = _mm_cvtepi32_ps(pixel_i_hi); // 4 floats
					__m128 mulled_hi = _mm_mul_ps(pixel_f_hi, mulfactor_dith_simd);
					converted32_hi = _mm_cvttps_epi32(mulled_hi); // truncate! rounding here makes brightness shift
				}
        else if constexpr(DITHER_BIT_DIFF > 0) {
					// round: nearest
					__m128 pixel_f_lo = _mm_cvtepi32_ps(pixel_i_lo); // 4 floats
					__m128 mulled_lo = _mm_mul_ps(pixel_f_lo, mulfactor_dith_simd);
					converted32_lo = _mm_cvtps_epi32(mulled_lo); // rounding ok, nearest. no +0.5 needed

					__m128 pixel_f_hi = _mm_cvtepi32_ps(pixel_i_hi); // 4 floats
					__m128 mulled_hi = _mm_mul_ps(pixel_f_hi, mulfactor_dith_simd);
					converted32_hi = _mm_cvtps_epi32(mulled_hi);
				}
				else {
					// new_pixel = (uint8_t)(srcp0[x] + corr);
					converted32_lo = pixel_i_lo;
					converted32_hi = pixel_i_hi;
				}

				__m128i converted_16 = _mm_packs_epi32(converted32_lo, converted32_hi);
        if constexpr(max_pixel_value_dithered <= 16384) // when <= 14 bits. otherwise packus_epi16 handles well. min_epi16 is sse2 only unlike min_epu16
					converted_16 = _mm_min_epi16(converted_16, _mm_set1_epi16(max_pixel_value_dithered)); // new_pixel = min(new_pixel, max_pixel_value_dithered); // clamp upper

																																																// scale back to the required bit depth
																																																// for generality. Now target == 8 bit, and dither_target is also 8 bit
																																																// for test: source:10 bit, target=8 bit, dither_target=4 bit
				const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
        if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0)  // dither to 8, target to 8
					converted_16 = _mm_slli_epi16(converted_16, BITDIFF_BETWEEN_DITHER_AND_TARGET); // new_pixel << BITDIFF_BETWEEN_DITHER_AND_TARGET; // if implemented non-8bit dither target, this should be fullscale

				__m128i converted_8 = _mm_packus_epi16(converted_16, zero);

				_mm_storel_epi64(reinterpret_cast<__m128i *>(&dstp[x]), converted_8);
			}
		} // x
		dstp += dst_pitch;
		srcp0 += src_pitch;
	}
}

// idea borrowed from fmtConv
#define FS_OPTIMIZED_SERPENTINE_COEF

template<int direction>
static RGY_FORCEINLINE void diffuse_floyd(int err, int &nextError, int *error_ptr)
{
#if defined (FS_OPTIMIZED_SERPENTINE_COEF)
	const int      e1 = 0;
	const int      e3 = (err * 4 + 8) >> 4;
#else
	const int      e1 = (err + 8) >> 4;
	const int      e3 = (err * 3 + 8) >> 4;
#endif
	const int      e5 = (err * 5 + 8) >> 4;
	const int      e7 = err - e1 - e3 - e5;

	nextError = error_ptr[direction];
	error_ptr[-direction] += e3;
	error_ptr[0] += e5;
	error_ptr[direction] = e1;
	nextError += e7;
}

template<int direction>
static void diffuse_floyd_f(float err, float &nextError, float *error_ptr)
{
#if defined (FS_OPTIMIZED_SERPENTINE_COEF)
	const float    e1 = 0;
	const float    e3 = err * (4.0f / 16);
#else
	const float    e1 = err * (1.0f / 16);
	const float    e3 = err * (3.0f / 16);
#endif
	const float    e5 = err * (5.0f / 16);
	const float    e7 = err * (7.0f / 16);

	nextError = error_ptr[direction];
	error_ptr[-direction] += e3;
	error_ptr[0] += e5;
	error_ptr[direction] = e1;
	nextError += e7;
}

template<typename source_pixel_t, typename target_pixel_t, uint8_t sourcebits, uint8_t TARGET_BITDEPTH, int TARGET_DITHER_BITDEPTH>
static void convert_uint_floyd_c(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const source_pixel_t *srcp = reinterpret_cast<const source_pixel_t *>(srcp8);
	src_pitch = src_pitch / sizeof(source_pixel_t);
	int src_width = src_rowsize / sizeof(source_pixel_t);

	target_pixel_t *dstp = reinterpret_cast<target_pixel_t *>(dstp8);
	dst_pitch = dst_pitch / sizeof(target_pixel_t);

	const int max_pixel_value = (1 << TARGET_BITDEPTH) - 1;
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);

	int *error_ptr_safe = new int[1 + src_width + 1]; // accumulated errors
	std::fill_n(error_ptr_safe, src_width + 2, 0);

	int *error_ptr = error_ptr_safe + 1;

	const int INTERNAL_BITS = DITHER_BIT_DIFF < 6 ? sourcebits + 8 : sourcebits; // keep accuracy
	const int SHIFTBITS_TO_INTERNAL = INTERNAL_BITS - sourcebits;
	const int SHIFTBITS_FROM_INTERNAL = INTERNAL_BITS - TARGET_DITHER_BITDEPTH;
	const int ROUNDER = 1 << (SHIFTBITS_FROM_INTERNAL - 1); // rounding

	for (int y = 0; y < src_height; y++)
	{
		int nextError = error_ptr[0];
		// serpentine forward
		if ((y & 1) == 0)
		{
			for (int x = 0; x < src_width; x++)
			{
				int err = nextError;
				int new_pixel = srcp[x] << SHIFTBITS_TO_INTERNAL; // if necessary
				int sum = new_pixel + err;
				int quantized = (sum + ROUNDER) >> (SHIFTBITS_FROM_INTERNAL);
				err = sum - (quantized << SHIFTBITS_FROM_INTERNAL);
				quantized <<= BITDIFF_BETWEEN_DITHER_AND_TARGET;
				int pix = max(min(max_pixel_value, quantized), 0); // clamp to target bit
				dstp[x] = (target_pixel_t)pix;
				diffuse_floyd<1>(err, nextError, error_ptr + x);
			}
		}
		else {
			// serpentine backward
			for (int x = src_width - 1; x >= 0; --x)
			{
				int err = nextError;
				int new_pixel = srcp[x] << SHIFTBITS_TO_INTERNAL; // if necessary
				int sum = new_pixel + err;
				int quantized = (sum + ROUNDER) >> (SHIFTBITS_FROM_INTERNAL);
				err = sum - (quantized << SHIFTBITS_FROM_INTERNAL);
				quantized <<= BITDIFF_BETWEEN_DITHER_AND_TARGET;
				int pix = max(min(max_pixel_value, quantized), 0); // clamp to target bit
				dstp[x] = (target_pixel_t)pix;
				diffuse_floyd<-1>(err, nextError, error_ptr + x);
			}
		}
		error_ptr[0] = nextError;
		dstp += dst_pitch;
		srcp += src_pitch;
	}

	delete[] error_ptr_safe;
}


// YUV conversions (bit shifts)
// BitDepthConvFuncPtr
// Conversion from 16-14-12-10 to 8 bits (bitshift: 8-6-4-2)
template<uint8_t sourcebits, int dither_mode, int TARGET_DITHER_BITDEPTH>
static void convert_uint16_to_8_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp0 = reinterpret_cast<const uint16_t *>(srcp);
	src_pitch = src_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);

	int _y = 0; // for ordered dither

	const int TARGET_BITDEPTH = 8; // here is constant (uint8_t target)
	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1;
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but generality)
	const int DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const int DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const int MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data); break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data); break;
	case 6: matrix = reinterpret_cast<const BYTE *>(dither8x8.data); break;
	case 8: matrix = reinterpret_cast<const BYTE *>(dither16x16.data); break;
  default: return; // n/a
	}

	for (int y = 0; y<src_height; y++)
	{
    if constexpr(dither_mode == 0) _y = (y & MASK) << DITHER_ORDER; // ordered dither
		for (int x = 0; x < src_width; x++)
		{
      if constexpr(dither_mode < 0) // -1: no dither
				dstp[x] = srcp0[x] >> (sourcebits - TARGET_BITDEPTH); // no dithering, no range conversion, simply shift
			else { // dither_mode == 0 -> ordered dither
				int corr = matrix[_y | (x & MASK)];
				//BYTE new_pixel = (((srcp0[x] << PRESHIFT) >> (sourcebits - 8)) + corr) >> PRESHIFT; // >> (sourcebits - 8);
				int new_pixel = ((srcp0[x] + corr) >> DITHER_BIT_DIFF);
				new_pixel = min(new_pixel, max_pixel_value_dithered); // clamp upper
																															// scale back to the required bit depth
																															// for generality. Now target == 8 bit, and dither_target is also 8 bit
																															// for test: source:10 bit, target=8 bit, dither_target=4 bit
				const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
        if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0)  // dither to 8, target to 8
					new_pixel = new_pixel << BITDIFF_BETWEEN_DITHER_AND_TARGET; // closest in palette: simple shift with
				dstp[x] = (BYTE)new_pixel;
			}
		}
		dstp += dst_pitch;
		srcp0 += src_pitch;
	}
}

template<uint8_t sourcebits>
static void convert_uint16_to_8_sse2(const BYTE *srcp8, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	src_pitch = src_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);
	int wmod16 = (src_width / 16) * 16;

	__m128i zero = _mm_setzero_si128();
	// no dithering, no range conversion, simply shift
	for (int y = 0; y < src_height; y++)
	{
		for (int x = 0; x < src_width; x += 16)
		{
			__m128i src_lo = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 8* uint16
			__m128i src_hi = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x + 8));
			src_lo = _mm_srli_epi16(src_lo, (sourcebits - 8));
			src_hi = _mm_srli_epi16(src_hi, (sourcebits - 8));
			__m128i dst = _mm_packus_epi16(src_lo, src_hi);
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), dst);
		}
		// rest
		for (int x = wmod16; x < src_width; x++)
		{
			dstp[x] = srcp[x] >> (sourcebits - 8);
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

#ifdef _MSC_VER
#pragma warning(push)
#pragma warning(disable: 4305 4309)
#endif

template<uint8_t sourcebits, uint8_t TARGET_DITHER_BITDEPTH>
static void convert_uint16_to_8_dither_sse2(const BYTE *srcp8, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	// Full general ordered dither from 10-16 bits to 2-8 bits, keeping the final 8 bit depth
	// Avisynth's ConvertBits parameter "dither_bits" default 8 goes to TARGET_DITHER_BITDEPTH
	// TARGET_BITDEPTH is always 8 bits, but the dither target can be less than 8.
	// The difference between source bitdepth and TARGET_DITHER_BITDEPTH cannot be more than 8
	// Basic usage: dither down to 8 bits from 10-16 bits.
	// Exotic usage: dither down to 2 bits from 10 bits

	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	src_pitch = src_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t); // real width. We take 2x8 word pixels at a time
	int wmod16 = (src_width / 16) * 16;

	int _y_c = 0; // Bayer matrix shift for ordered dither

	const uint8_t TARGET_BITDEPTH = 8; // here is constant (uint8_t target)
	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1; //may be less than 255, e.g. 15 for dither target 4 bits

	const __m128i max_pixel_value_dithered_epi8 = _mm_set1_epi8(max_pixel_value_dithered);
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const uint8_t DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const uint8_t PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but for the sake of generality)
	const uint8_t DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const uint8_t DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const uint8_t MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	const BYTE *matrix_c;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data_sse2);
		matrix_c = reinterpret_cast<const BYTE *>(dither2x2.data);
		break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data_sse2);
		matrix_c = reinterpret_cast<const BYTE *>(dither4x4.data);
		break;
	case 6:
		matrix = reinterpret_cast<const BYTE *>(dither8x8.data_sse2);
		matrix_c = reinterpret_cast<const BYTE *>(dither8x8.data);
		break;
	case 8:
		matrix = reinterpret_cast<const BYTE *>(dither16x16.data);
		matrix_c = matrix;
		break;
  default: return; // n/a
	}

	const BYTE *current_matrix_line;

	__m128i zero = _mm_setzero_si128();

	for (int y = 0; y < src_height; y++)
	{
		_y_c = (y & MASK) << DITHER_ORDER; // matrix lines stride for C
		current_matrix_line = matrix + ((y & MASK) << 4); // always 16 byte boundary

		__m128i corr = _mm_load_si128(reinterpret_cast<const __m128i*>(current_matrix_line)); // int corr = matrix[_y | (x & MASK)];
		__m128i corr_lo = _mm_unpacklo_epi8(corr, zero); // lower 8 byte->uint16_t
		__m128i corr_hi = _mm_unpackhi_epi8(corr, zero); // upper 8 byte->uint16_t

		for (int x = 0; x < src_width; x += 16)
		{
			__m128i src_lo = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 8* uint16
			__m128i src_hi = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x + 8));

			// int new_pixel = ((srcp0[x] + corr) >> DITHER_BIT_DIFF);

			__m128i new_pixel_lo, new_pixel_hi;

      if constexpr(sourcebits < 16) { // no overflow
				new_pixel_lo = _mm_srli_epi16(_mm_add_epi16(src_lo, corr_lo), DITHER_BIT_DIFF);
				new_pixel_hi = _mm_srli_epi16(_mm_add_epi16(src_hi, corr_hi), DITHER_BIT_DIFF);
				// scale down after adding dithering noise
			}
			else { // source bits: 16. Overflow can happen when 0xFFFF it dithered up. Go 32 bits
						 // lower
				__m128i src_lo_lo = _mm_unpacklo_epi16(src_lo, zero);
				__m128i corr_lo_lo = _mm_unpacklo_epi16(corr_lo, zero);
				__m128i new_pixel_lo_lo = _mm_srli_epi32(_mm_add_epi32(src_lo_lo, corr_lo_lo), DITHER_BIT_DIFF);

				__m128i src_lo_hi = _mm_unpackhi_epi16(src_lo, zero);
				__m128i corr_lo_hi = _mm_unpackhi_epi16(corr_lo, zero);
				__m128i new_pixel_lo_hi = _mm_srli_epi32(_mm_add_epi32(src_lo_hi, corr_lo_hi), DITHER_BIT_DIFF);

				new_pixel_lo = _mm_packs_epi32(new_pixel_lo_lo, new_pixel_lo_hi); // packs is enough
																																					// upper
				__m128i src_hi_lo = _mm_unpacklo_epi16(src_hi, zero);
				__m128i corr_hi_lo = _mm_unpacklo_epi16(corr_hi, zero);
				__m128i new_pixel_hi_lo = _mm_srli_epi32(_mm_add_epi32(src_hi_lo, corr_hi_lo), DITHER_BIT_DIFF);

				__m128i src_hi_hi = _mm_unpackhi_epi16(src_hi, zero);
				__m128i corr_hi_hi = _mm_unpackhi_epi16(corr_hi, zero);
				__m128i new_pixel_hi_hi = _mm_srli_epi32(_mm_add_epi32(src_hi_hi, corr_hi_hi), DITHER_BIT_DIFF);

				new_pixel_hi = _mm_packs_epi32(new_pixel_hi_lo, new_pixel_hi_hi); // packs is enough
			}

			__m128i new_pixel = _mm_packus_epi16(new_pixel_lo, new_pixel_hi); // 2x8 x16 bit -> 16 byte. Clamp is automatic

      if constexpr(TARGET_DITHER_BITDEPTH < 8) { // generic (not used) fun option to dither 10->4 bits then back to 8 bit
				new_pixel = _mm_min_epu8(new_pixel, max_pixel_value_dithered_epi8);
			}

			const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
      if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0) { //==0 when dither and target are both 8
																										// scale back, when e.g. 10 bit data is dithered down to 4,6,8 bits but the target bit depth is still 8 bit.
				new_pixel = _mm_and_si128(_mm_set1_epi8((0xFF << BITDIFF_BETWEEN_DITHER_AND_TARGET) & 0xFF), _mm_slli_epi32(new_pixel, BITDIFF_BETWEEN_DITHER_AND_TARGET));
				// non-existant _mm_slli_epi8. closest in palette: simple shift
			}

			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), new_pixel);
		}

		// rest, C
		for (int x = wmod16; x < src_width; x++)
		{
			int corr = matrix_c[_y_c | (x & MASK)];
			//BYTE new_pixel = (((srcp0[x] << PRESHIFT) >> (sourcebits - 8)) + corr) >> PRESHIFT; // >> (sourcebits - 8);
			int new_pixel = ((srcp[x] + corr) >> DITHER_BIT_DIFF);
			new_pixel = min(new_pixel, max_pixel_value_dithered); // clamp upper
																														// scale back to the required bit depth
																														// for generality. Now target == 8 bit, and dither_target is also 8 bit
																														// for test: source:10 bit, target=8 bit, dither_target=4 bit
			const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
      if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0)  // dither to 8, target to 8
				new_pixel = new_pixel << BITDIFF_BETWEEN_DITHER_AND_TARGET; // closest in palette: simple shift with
			dstp[x] = (BYTE)new_pixel;
		}

		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

// 10-16bits: sse4.1
// 8 bits: sse2
template<typename pixel_t, uint8_t targetbits, bool chroma, bool fulls, bool fulld>
void convert_32_to_uintN_sse(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const float *srcp = reinterpret_cast<const float *>(srcp8);
	pixel_t *dstp = reinterpret_cast<pixel_t *>(dstp8);

	src_pitch = src_pitch / sizeof(float);
	dst_pitch = dst_pitch / sizeof(pixel_t);

	int src_width = src_rowsize / sizeof(float);

	const int max_pixel_value = (1 << targetbits) - 1;
	const __m128i max_pixel_value_128 = _mm_set1_epi16(max_pixel_value);

  const int limit_lo_d = (fulld ? 0 : 16) << (targetbits - 8);
  const int limit_hi_d = fulld ? ((1 << targetbits) - 1) : ((chroma ? 240 : 235) << (targetbits - 8));
  const float range_diff_d = (float)limit_hi_d - limit_lo_d;

  const int limit_lo_s = fulls ? 0 : 16;
  const int limit_hi_s = fulls ? 255 : (chroma ? 240 : 235);
  const float range_diff_s = (limit_hi_s - limit_lo_s) / 255.0f;

  // fulls fulld luma             luma_new   chroma                          chroma_new
  // true  false 0..1              16-235     -0.5..0.5                      16-240       Y = Y * ((235-16) << (bpp-8)) + 16, Chroma= Chroma * ((240-16) << (bpp-8)) + 16
  // true  true  0..1               0-255     -0.5..0.5                      0-128-255
  // false false 16/255..235/255   16-235     (16-128)/255..(240-128)/255    16-240
  // false true  16/255..235/255    0..1      (16-128)/255..(240-128)/255    0-128-255
  const float factor = range_diff_d / range_diff_s;

	const float half_i = (float)(1 << (targetbits - 1));
#ifdef FLOAT_CHROMA_IS_HALF_CENTERED
	const __m128 half_ps = _mm_set1_ps(0.5f);
#endif
	const __m128 halfint_plus_rounder_ps = _mm_set1_ps(half_i + 0.5f);
  const __m128 limit_lo_s_ps = _mm_set1_ps(limit_lo_s / 255.0f);
  const __m128 limit_lo_plus_rounder_ps = _mm_set1_ps(limit_lo_d + 0.5f);

  __m128 factor_ps = _mm_set1_ps(factor); // 0-1.0 -> 0..max_pixel_value

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x += 8) // 8 pixels at a time
		{
			__m128i result;
			__m128i result_0, result_1;
			__m128 src_0 = _mm_load_ps(reinterpret_cast<const float *>(srcp + x));
			__m128 src_1 = _mm_load_ps(reinterpret_cast<const float *>(srcp + x + 4));
			if (chroma) {
#ifdef FLOAT_CHROMA_IS_HALF_CENTERED
				// shift 0.5 before, shift back half_int after. 0.5->exact half of 128/512/...
				src_0 = _mm_sub_ps(src_0, half_ps);
				src_1 = _mm_sub_ps(src_1, half_ps);
				//pixel = (srcp0[x] - 0.5f) * factor + half + 0.5f;
#else
        //pixel = (srcp0[x]       ) * factor + half + 0.5f;
#endif
				src_0 = _mm_add_ps(_mm_mul_ps(src_0, factor_ps), halfint_plus_rounder_ps);
				src_1 = _mm_add_ps(_mm_mul_ps(src_1, factor_ps), halfint_plus_rounder_ps);
			}
			else {
        if constexpr(!fulls) {
          src_0 = _mm_sub_ps(src_0, limit_lo_s_ps);
          src_1 = _mm_sub_ps(src_1, limit_lo_s_ps);
        }
        src_0 = _mm_add_ps(_mm_mul_ps(src_0, factor_ps), limit_lo_plus_rounder_ps);
        src_1 = _mm_add_ps(_mm_mul_ps(src_1, factor_ps), limit_lo_plus_rounder_ps);
        //pixel = (srcp0[x] - limit_lo_s_ps) * factor + half + limit_lo + 0.5f;
			}
			result_0 = _mm_cvttps_epi32(src_0); // truncate
			result_1 = _mm_cvttps_epi32(src_1);
      if constexpr(sizeof(pixel_t) == 2) {
				result = af_mm_packus_epi32<true>(result_0, result_1); // sse41
        if constexpr(targetbits > 8 && targetbits < 16) {
					result = af_mm_min_epu16<true>(result, max_pixel_value_128); // sse41, extra clamp for 10, 12, 14 bits
				}
				_mm_store_si128(reinterpret_cast<__m128i *>(dstp + x), result);
			}
			else {
				result = af_mm_packus_epi32<sizeof(pixel_t) == 2>(result_0, result_1);
				result = _mm_packus_epi16(result, result); // lo 8 byte
				_mm_storel_epi64(reinterpret_cast<__m128i *>(dstp + x), result);
			}
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}
#ifdef _MSC_VER
#pragma warning(pop)
#endif

// float to 8 bit, float to 10/12/14/16 bit
template<typename pixel_t, uint8_t targetbits, bool chroma, bool fulls, bool fulld>
static void convert_32_to_uintN_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const float *srcp0 = reinterpret_cast<const float *>(srcp);
	pixel_t *dstp0 = reinterpret_cast<pixel_t *>(dstp);

	src_pitch = src_pitch / sizeof(float);
	dst_pitch = dst_pitch / sizeof(pixel_t);

	int src_width = src_rowsize / sizeof(float);

	const float max_dst_pixelvalue = (float)((1 << targetbits) - 1); // 255, 1023, 4095, 16383, 65535.0
	const float half = (float)(1 << (targetbits - 1));

  const int limit_lo_d = (fulld ? 0 : 16) << (targetbits - 8);
  const int limit_hi_d = fulld ? ((1 << targetbits) - 1) : ((chroma ? 240 : 235) << (targetbits - 8));
  const float range_diff_d = (float)limit_hi_d - limit_lo_d;

  const int limit_lo_s = fulls ? 0 : 16;
  const float limit_lo_s_ps = limit_lo_s / 255.0f;
  const int limit_hi_s = fulls ? 255 : (chroma ? 240 : 235);
  const float range_diff_s = (limit_hi_s - limit_lo_s) / 255.0f; 

  // fulls fulld luma             luma_new   chroma                          chroma_new
  // true  false 0..1              16-235     -0.5..0.5                      16-240       Y = Y * ((235-16) << (bpp-8)) + 16, Chroma= Chroma * ((240-16) << (bpp-8)) + 16
  // true  true  0..1               0-255     -0.5..0.5                      0-128-255
  // false false 16/255..235/255   16-235     (16-128)/255..(240-128)/255    16-240
  // false true  16/255..235/255    0..1      (16-128)/255..(240-128)/255    0-128-255
  const float factor = range_diff_d / range_diff_s;

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			float pixel;
			if (chroma) {
#ifdef FLOAT_CHROMA_IS_HALF_CENTERED
				// shift 0.5 before, shift back half_int after. 0.5->exact half of 128/512/...
        pixel = (srcp0[x] - 0.5f);
#else
        pixel = srcp0[x];
#endif
        pixel = pixel * factor + half + 0.5f;
			}
			else {
        if constexpr(!fulls)
          pixel = (srcp0[x] - limit_lo_s_ps) * factor + 0.5f + limit_lo_d;
        else
          pixel = srcp0[x] * factor + 0.5f + limit_lo_d;
			}
			dstp0[x] = pixel_t(clamp(pixel, 0.0f, max_dst_pixelvalue)); // we clamp here!
		}
		dstp0 += dst_pitch;
		srcp0 += src_pitch;
	}
}

// rgb/alpha: full scale. No bit shift, scale full ranges
template<uint8_t targetbits>
static void convert_rgb_8_to_uint16_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint8_t *srcp0 = reinterpret_cast<const uint8_t *>(srcp);
	uint16_t *dstp0 = reinterpret_cast<uint16_t *>(dstp);

	src_pitch = src_pitch / sizeof(uint8_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	int src_width = src_rowsize / sizeof(uint8_t);

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			// test
            if constexpr(targetbits==16)
				dstp0[x] = srcp0[x] * 257; // full range 0..255 <-> 0..65535 (257 = 65535 / 255)
            else if constexpr(targetbits==14)
				dstp0[x] = srcp0[x] * 16383 / 255; // full range 0..255 <-> 0..16384-1
            else if constexpr(targetbits==12)
				dstp0[x] = srcp0[x] * 4095 / 255; // full range 0..255 <-> 0..4096-1
            else if constexpr(targetbits==10)
				dstp0[x] = srcp0[x] * 1023 / 255; // full range 0..255 <-> 0..1024-1
		}
		dstp0 += dst_pitch;
		srcp0 += src_pitch;
	}
}

#if 0
// leave it here, maybe we can use it later
// Tricky simd implementation of integer div 255 w/o division
static inline __m128i Div_4xint32_by_255(const __m128i &esi, const __m128i &magic255div) {
	// simd implementation of
	/*
	Trick of integer/255 w/o division:
	tmp = (int)((( (int64_t)esi * (-2139062143)) >> 32) & 0xFFFFFFFF) + esi) >> 7
	result = tmp + (tmp >> 31)

	movzx	eax, BYTE PTR [ecx+edi] // orig pixel
	imul	esi, eax, 16383         // * Scale_Multiplier
	// div 255 follows
	// result in esi is int32
	// Div_4xint32_by_255 implementation from here!
	mov	eax, -2139062143			; 80808081H
	imul	esi  // signed!
	add	edx, esi
	sar	edx, 7
	mov	eax, edx
	shr	eax, 31					; 0000001fH
	add	eax, edx
	mov	WORD PTR [ebx+ecx*2], ax
	*/
	// edx_eax_64 = mulres_lo(esi) * magic255div(eax)
	// _mm_mul_epu32: r64_0 := a0 * b0, r64_1 := a2 * b2 (edx_eax edx_eax)
	// signed mul!
	__m128i mulwithmagic02 = _mm_mul_epi32(esi, magic255div); // signed! need epi not epu! only sse4.1
	__m128i mulwithmagic13 = _mm_mul_epi32(_mm_srli_si128(esi, 4), magic255div);
	// shuffle hi32bit of results to [63..0] and pack. a3->a1, a1->a0
	__m128i upper32bits_edx = _mm_unpacklo_epi32(_mm_shuffle_epi32(mulwithmagic02, _MM_SHUFFLE(0, 0, 3, 1)), _mm_shuffle_epi32(mulwithmagic13, _MM_SHUFFLE(0, 0, 3, 1)));

	// vvv lower 32 bit of result is never used in the algorithm
	// shuffle lo32bit results to [63..0] and pack
	// __m128i lower32bits_eax = _mm_unpacklo_epi32(_mm_shuffle_epi32(mulwithmagic02, _MM_SHUFFLE (0,0,2,0)), _mm_shuffle_epi32(mulwithmagic13, _MM_SHUFFLE (0,0,2,0)));

	// add edx, mulres_lo(esi)
	__m128i tmp_edx = _mm_add_epi32(upper32bits_edx, esi);
	// sar edx, 7
	// shift arithmetic
	tmp_edx = _mm_srai_epi32(tmp_edx, 7);
	// mov eax, edx
	// shr eax, 31					; 0000001fH
	// shift logical
	__m128i tmp_eax = _mm_srli_epi32(tmp_edx, 31);
	// add eax, edx
	__m128i result = _mm_add_epi32(tmp_eax, tmp_edx);
	return result;
	// 4 results in the lower 16 bits of 4x32 bit register
}
#endif

template<uint8_t targetbits>
static void convert_rgb_8_to_uint16_sse2(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint8_t *srcp = reinterpret_cast<const uint8_t *>(srcp8);
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);

	src_pitch = src_pitch / sizeof(uint8_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	int src_width = src_rowsize / sizeof(uint8_t);
	int wmod16 = (src_width / 16) * 16;

	const int MUL = (targetbits == 16) ? 257 : ((1 << targetbits) - 1);
	const int DIV = (targetbits == 16) ? 1 : 255;
	// 16 bit: one mul only, no need for /255
	// for others: // *16383 *4095 *1023  and /255

	__m128i zero = _mm_setzero_si128();
	__m128i multiplier = _mm_set1_epi16(MUL);
	__m128i magic255div = _mm_set1_epi32(-2139062143); // 80808081H
	__m128 multiplier_float = _mm_set1_ps((float)MUL / DIV);
	// This is ok, since the default SIMD rounding mode is round-to-nearest unlike c++ truncate
	// in C: 1023 * multiplier = 1022.999 -> truncates.

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x += 16)
		{
			__m128i src = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 16* uint8
			__m128i src_lo = _mm_unpacklo_epi8(src, zero);             // 8* uint16
			__m128i src_hi = _mm_unpackhi_epi8(src, zero);             // 8* uint16
																																 // test
      if constexpr(targetbits==16) {
				// *257 mullo is faster than x*257 = (x<<8 + x) add/or solution (i7)
				__m128i res_lo = _mm_mullo_epi16(src_lo, multiplier); // lower 16 bit of multiplication is enough
				__m128i res_hi = _mm_mullo_epi16(src_hi, multiplier);
				// dstp[x] = srcp[x] * 257; // RGB: full range 0..255 <-> 0..65535 (257 = 65535 / 255)
				_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), res_lo);
				_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x + 8), res_hi);
			}
			else {
#if 0
				if (false) {
					// dead end
					// simulate integer tricky div 255 arithmetic.
					// Unfortunately it's sse41 only plus much slower than float, but still much faster than C. Too much overhead

					// process 8*uint16_t
					//--------------
					// first src_lo

					// imul	esi, eax, 16383
					__m128i res_lower16bit = _mm_mullo_epi16(src_lo, multiplier); // *16383 *4095 *1023 result: int32. get lower 16
					__m128i res_upper16bit = _mm_mulhi_epi16(src_lo, multiplier); // *16383 *4095 *1023 result: int32. get upper 16
					__m128i mulres_lo = _mm_unpacklo_epi16(res_lower16bit, res_upper16bit); // 4 int32
					__m128i mulres_hi = _mm_unpackhi_epi16(res_lower16bit, res_upper16bit); // 4 int32

																																									// process first 4 of 8 uint32_t (mulres_lo)
					__m128i tmp_eax_lo = Div_4xint32_by_255(mulres_lo, magic255div);
					// process second 4 of 8 uint32_t (mulres_hi)
					__m128i tmp_eax_hi = Div_4xint32_by_255(mulres_hi, magic255div);
					__m128i dst = _mm_packus_epi32(tmp_eax_lo, tmp_eax_hi);
					_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), dst);

					//--------------
					// second src_hi
					{
						// imul	esi, eax, 16383|4095|1023
						__m128i res_lower16bit = _mm_mullo_epi16(src_hi, multiplier); // *16383 *4095 *1023 result: int32. get lower 16
						__m128i res_upper16bit = _mm_mulhi_epi16(src_hi, multiplier); // *16383 *4095 *1023 result: int32. get upper 16
						__m128i mulres_lo = _mm_unpacklo_epi16(res_lower16bit, res_upper16bit); // 4 int32
						__m128i mulres_hi = _mm_unpackhi_epi16(res_lower16bit, res_upper16bit); // 4 int32

																																										// process first 4 of 8 uint32_t (mulres_lo)
						__m128i tmp_eax_lo = Div_4xint32_by_255(mulres_lo, magic255div);
						// process second 4 of 8 uint32_t (mulres_hi)
						__m128i tmp_eax_hi = Div_4xint32_by_255(mulres_hi, magic255div);
						__m128i dst = _mm_packus_epi32(tmp_eax_lo, tmp_eax_hi);
						_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x + 8), dst);
					}
				}
				else
#endif
				{
					// src_lo: 8*uint16
					// convert to int32 then float, multiply and convert back
					__m128 res_lo = _mm_mul_ps(_mm_cvtepi32_ps(_mm_unpacklo_epi16(src_lo, zero)), multiplier_float);
					__m128 res_hi = _mm_mul_ps(_mm_cvtepi32_ps(_mm_unpackhi_epi16(src_lo, zero)), multiplier_float);
					// Converts the four single-precision, floating-point values of a to signed 32-bit integer values.
					__m128i result_l = _mm_cvtps_epi32(res_lo); // The default rounding mode is round-to-nearest unlike c++ truncate
					__m128i result_h = _mm_cvtps_epi32(res_hi);
					// Pack and store no need for packus for <= 14 bit
					__m128i result = _mm_packs_epi32(result_l, result_h); // 4*32+4*32 = 8*16
					_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), result);

					// src_hi: 8*uint16
					// convert to int32 then float, multiply and convert back
					res_lo = _mm_mul_ps(_mm_cvtepi32_ps(_mm_unpacklo_epi16(src_hi, zero)), multiplier_float);
					res_hi = _mm_mul_ps(_mm_cvtepi32_ps(_mm_unpackhi_epi16(src_hi, zero)), multiplier_float);
					// Converts the four single-precision, floating-point values of a to signed 32-bit integer values.
					result_l = _mm_cvtps_epi32(res_lo);
					result_h = _mm_cvtps_epi32(res_hi);
					// Pack and store no need for packus for <= 14 bit
					result = _mm_packs_epi32(result_l, result_h); // 4*32+4*32 = 8*16
					_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x + 8), result);
				}
			} // if 16 bit else
		} // for x
			// rest
		for (int x = wmod16; x < src_width; x++)
		{
			dstp[x] = srcp[x] * MUL / DIV; // RGB: full range 0..255 <-> 0..16384-1
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	} // for y
}


// YUV: bit shift 8 to 10-12-14-16 bits
template<uint8_t targetbits>
static void convert_8_to_uint16_c(const BYTE *srcp, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);

	dst_pitch = dst_pitch / sizeof(uint16_t);

	int src_width = src_rowsize / sizeof(uint8_t); // intentional

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			dstp[x] = srcp[x] << (targetbits - 8);
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

template<uint8_t targetbits>
static void convert_8_to_uint16_sse2(const BYTE *srcp, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);

	dst_pitch = dst_pitch / sizeof(uint16_t);

	int src_width = src_rowsize / sizeof(uint8_t);
	int wmod16 = (src_width / 16) * 16;

	__m128i zero = _mm_setzero_si128();

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x += 16)
		{
			__m128i src = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 16 bytes
			__m128i dst_lo = _mm_unpacklo_epi8(src, zero);
			__m128i dst_hi = _mm_unpackhi_epi8(src, zero);
			dst_lo = _mm_slli_epi16(dst_lo, (targetbits - 8));
			dst_hi = _mm_slli_epi16(dst_hi, (targetbits - 8));
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), dst_lo);
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x + 8), dst_hi);
		}
		// rest
		for (int x = wmod16; x < src_width; x++)
		{
			dstp[x] = srcp[x] << (targetbits - 8);
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

// RGB full range: 10-12-14-16 <=> 10-12-14-16 bits
template<uint8_t sourcebits, uint8_t targetbits, bool hasSSE4>
static void convert_rgb_uint16_to_uint16_sse2(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	src_pitch = src_pitch / sizeof(uint16_t);
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);
	dst_pitch = dst_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);
	int wmod = (src_width / 8) * 8;

	const uint16_t source_max = (1 << sourcebits) - 1;
	const uint16_t target_max = (1 << targetbits) - 1;

	__m128 factor = _mm_set1_ps((float)target_max / source_max);
	__m128i max_pixel_value = _mm_set1_epi16(target_max);
	__m128i zero = _mm_setzero_si128();

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x += 8)
		{
			__m128i src = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 8* uint16

			__m128i src_lo = _mm_unpacklo_epi16(src, zero);
			__m128i src_hi = _mm_unpackhi_epi16(src, zero);

			__m128 result_lo = _mm_mul_ps(_mm_cvtepi32_ps(src_lo), factor);
			__m128 result_hi = _mm_mul_ps(_mm_cvtepi32_ps(src_hi), factor);

			__m128i result = af_mm_packus_epi32<hasSSE4>(_mm_cvtps_epi32(result_lo), _mm_cvtps_epi32(result_hi));
			if constexpr(targetbits < 16) {
				result = af_mm_min_epu16<hasSSE4>(result, max_pixel_value);
			}
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), result);
		}
		// rest
		for (int x = wmod; x < src_width; x++)
		{
			dstp[x] = (uint16_t)((int64_t)srcp[x] * target_max / source_max); // expand range
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}


// RGB full range: 10-12-14-16 <=> 10-12-14-16 bits
template<uint8_t sourcebits, uint8_t targetbits>
static void convert_rgb_uint16_to_uint16_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp0 = reinterpret_cast<const uint16_t *>(srcp);
	uint16_t *dstp0 = reinterpret_cast<uint16_t *>(dstp);

	src_pitch = src_pitch / sizeof(uint16_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	const int src_width = src_rowsize / sizeof(uint16_t);

	const uint16_t source_max = (1 << sourcebits) - 1;
	const uint16_t target_max = (1 << targetbits) - 1;

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			// int64: avoid unsigned * unsigned = signed arithmetic overflow
			dstp0[x] = (uint16_t)((int64_t)srcp0[x] * target_max / source_max);
		}
		dstp0 += dst_pitch;
		srcp0 += src_pitch;
	}
}

template<uint8_t sourcebits, uint8_t targetbits, int TARGET_DITHER_BITDEPTH>
static void convert_rgb_uint16_to_uint16_dither_c(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);

	src_pitch = src_pitch / sizeof(uint16_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	const int src_width = src_rowsize / sizeof(uint16_t);

	const int source_max = (1 << sourcebits) - 1;

	int _y = 0; // for ordered dither

	const int TARGET_BITDEPTH = targetbits;
	const int max_pixel_value = (1 << TARGET_BITDEPTH) - 1;
	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1;
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but generality)
	const int DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const int DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const int MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data); break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data); break;
	case 6: matrix = reinterpret_cast<const BYTE *>(dither8x8.data); break;
	case 8: matrix = reinterpret_cast<const BYTE *>(dither16x16.data); break;
  default: return; // n/a
	}

	for (int y = 0; y < src_height; y++)
	{
		_y = (y & MASK) << DITHER_ORDER; // ordered dither
		for (int x = 0; x < src_width; x++)
		{
			int corr = matrix[_y | (x & MASK)];
			//BYTE new_pixel = (((srcp0[x] << PRESHIFT) >> (sourcebits - 8)) + corr) >> PRESHIFT; // >> (sourcebits - 8);
			//int new_pixel = ((srcp[x] + corr) >> DITHER_BIT_DIFF);
			int64_t new_pixel = (int64_t)(srcp[x] + corr) * max_pixel_value_dithered / source_max;

			// new_pixel = min(new_pixel, max_pixel_value_dithered_i); // clamp upper
			// scale back to the required bit depth
			const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
      if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0) {
				new_pixel = new_pixel * max_pixel_value / max_pixel_value_dithered;
			}
			dstp[x] = (uint16_t)(min((int)new_pixel, max_pixel_value));
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}

}

// YUV: bit shift 10-12-14-16 <=> 10-12-14-16 bits
// shift right or left, depending on expandrange template param
template<bool expandrange, uint8_t shiftbits>
static void convert_uint16_to_uint16_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp0 = reinterpret_cast<const uint16_t *>(srcp);
	uint16_t *dstp0 = reinterpret_cast<uint16_t *>(dstp);

	src_pitch = src_pitch / sizeof(uint16_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	const int src_width = src_rowsize / sizeof(uint16_t);

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			if (expandrange)
				dstp0[x] = srcp0[x] << shiftbits;  // expand range. No clamp before, source is assumed to have valid range
			else
				dstp0[x] = srcp0[x] >> shiftbits;  // reduce range
		}
		dstp0 += dst_pitch;
		srcp0 += src_pitch;
	}
}

// YUV: bit shift 10-12-14-16 <=> 10-12-14-16 bits
// shift right or left, depending on expandrange template param
template<uint8_t sourcebits, uint8_t targetbits, int TARGET_DITHER_BITDEPTH>
static void convert_uint16_to_uint16_dither_c(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);

	src_pitch = src_pitch / sizeof(uint16_t);
	dst_pitch = dst_pitch / sizeof(uint16_t);

	const int src_width = src_rowsize / sizeof(uint16_t);

	int _y = 0; // for ordered dither

	const int TARGET_BITDEPTH = targetbits;
	const int max_pixel_value_dithered = (1 << TARGET_DITHER_BITDEPTH) - 1;
	// precheck ensures:
	// TARGET_BITDEPTH >= TARGET_DITHER_BITDEPTH
	// sourcebits - TARGET_DITHER_BITDEPTH <= 8
	// sourcebits - TARGET_DITHER_BITDEPTH is even (later we can use PRESHIFT)
	const int DITHER_BIT_DIFF = (sourcebits - TARGET_DITHER_BITDEPTH); // 2, 4, 6, 8
	const int PRESHIFT = DITHER_BIT_DIFF & 1;  // 0 or 1: correction for odd bit differences (not used here but generality)
	const int DITHER_ORDER = (DITHER_BIT_DIFF + PRESHIFT) / 2;
	const int DITHER_SIZE = 1 << DITHER_ORDER; // 9,10=2  11,12=4  13,14=8  15,16=16
	const int MASK = DITHER_SIZE - 1;
	// 10->8: 0x01 (2x2)
	// 11->8: 0x03 (4x4)
	// 12->8: 0x03 (4x4)
	// 14->8: 0x07 (8x8)
	// 16->8: 0x0F (16x16)
	const BYTE *matrix;
	switch (sourcebits - TARGET_DITHER_BITDEPTH) {
	case 2: matrix = reinterpret_cast<const BYTE *>(dither2x2.data); break;
	case 4: matrix = reinterpret_cast<const BYTE *>(dither4x4.data); break;
	case 6: matrix = reinterpret_cast<const BYTE *>(dither8x8.data); break;
	case 8: matrix = reinterpret_cast<const BYTE *>(dither16x16.data); break;
  default: return; // n/a
	}

	for (int y = 0; y < src_height; y++)
	{
		_y = (y & MASK) << DITHER_ORDER; // ordered dither
		for (int x = 0; x < src_width; x++)
		{
			int corr = matrix[_y | (x & MASK)];
			//BYTE new_pixel = (((srcp0[x] << PRESHIFT) >> (sourcebits - 8)) + corr) >> PRESHIFT; // >> (sourcebits - 8);
			int new_pixel = ((srcp[x] + corr) >> DITHER_BIT_DIFF);
			new_pixel = min(new_pixel, max_pixel_value_dithered); // clamp upper
																														// scale back to the required bit depth
																														// for generality. Now target == 8 bit, and dither_target is also 8 bit
																														// for test: source:10 bit, target=8 bit, dither_target=4 bit
			const int BITDIFF_BETWEEN_DITHER_AND_TARGET = DITHER_BIT_DIFF - (sourcebits - TARGET_BITDEPTH);
      if constexpr(BITDIFF_BETWEEN_DITHER_AND_TARGET != 0)  // dither to 8, target to 8
				new_pixel = new_pixel << BITDIFF_BETWEEN_DITHER_AND_TARGET; // closest in palette: simple shift with
			dstp[x] = (uint16_t)new_pixel;
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

template<bool expandrange, uint8_t shiftbits>
static void convert_uint16_to_uint16_sse2(const BYTE *srcp8, BYTE *dstp8, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	// remark: Compiler with SSE2 option generates the same effective code like this in C
	// Drawback of SSE2: a future avx2 target gives more efficient code than inline SSE2 (256 bit registers)
	const uint16_t *srcp = reinterpret_cast<const uint16_t *>(srcp8);
	src_pitch = src_pitch / sizeof(uint16_t);
	uint16_t *dstp = reinterpret_cast<uint16_t *>(dstp8);
	dst_pitch = dst_pitch / sizeof(uint16_t);
	int src_width = src_rowsize / sizeof(uint16_t);
	int wmod = (src_width / 16) * 16;

	__m128i shift = _mm_set_epi32(0, 0, 0, shiftbits);

	// no dithering, no range conversion, simply shift
	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x += 16)
		{
			__m128i src_lo = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x)); // 8* uint16
			__m128i src_hi = _mm_load_si128(reinterpret_cast<const __m128i*>(srcp + x + 8)); // 8* uint16
			if (expandrange) {
				src_lo = _mm_sll_epi16(src_lo, shift);
				src_hi = _mm_sll_epi16(src_hi, shift);
			}
			else {
				src_lo = _mm_srl_epi16(src_lo, shift);
				src_hi = _mm_srl_epi16(src_hi, shift);
			}
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x), src_lo);
			_mm_store_si128(reinterpret_cast<__m128i*>(dstp + x + 8), src_hi);
		}
		// rest
		for (int x = wmod; x < src_width; x++)
		{
			if (expandrange)
				dstp[x] = srcp[x] << shiftbits;  // expand range. No clamp before, source is assumed to have valid range
			else
				dstp[x] = srcp[x] >> shiftbits;  // reduce range
		}
		dstp += dst_pitch;
		srcp += src_pitch;
	}
}

// 8 bit to float, 16/14/12/10 bits to float
template<typename pixel_t, uint8_t sourcebits, bool chroma, bool fulls, bool fulld>
static void convert_uintN_to_float_c(const BYTE *srcp, BYTE *dstp, int src_rowsize, int src_height, int src_pitch, int dst_pitch)
{
	const pixel_t *srcp0 = reinterpret_cast<const pixel_t *>(srcp);
	float *dstp0 = reinterpret_cast<float *>(dstp);

	src_pitch = src_pitch / sizeof(pixel_t);
	dst_pitch = dst_pitch / sizeof(float);

	int src_width = src_rowsize / sizeof(pixel_t);

  const int limit_lo_s = (fulls ? 0 : 16) << (sourcebits - 8);
  const int limit_hi_s = fulls ? ((1 << sourcebits) - 1) : ((chroma ? 240 : 235) << (sourcebits - 8));
  const float range_diff_s = (float)limit_hi_s - limit_lo_s;

  const int limit_lo_d = fulld ? 0 : 16;
  const int limit_hi_d = fulld ? 255 : (chroma ? 240 : 235);
  const float range_diff_d = (limit_hi_d - limit_lo_d) / 255.0f;

  // fulls fulld luma             luma_new   chroma                          chroma_new
  // true  false 0..1              16-235     -0.5..0.5                      16-240       Y = Y * ((235-16) << (bpp-8)) + 16, Chroma= Chroma * ((240-16) << (bpp-8)) + 16
  // true  true  0..1               0-255     -0.5..0.5                      0-128-255
  // false false 16/255..235/255   16-235     (16-128)/255..(240-128)/255    16-240
  // false true  16/255..235/255    0..1      (16-128)/255..(240-128)/255    0-128-255
  const float factor = range_diff_d / range_diff_s;

  const int half = 1 << (sourcebits - 1);

  // 0..255,65535 -> 0..1.0 (or -0.5..+0.5) or less if !full

	for (int y = 0; y<src_height; y++)
	{
		for (int x = 0; x < src_width; x++)
		{
			float pixel;
			if (chroma) {
#ifdef FLOAT_CHROMA_IS_HALF_CENTERED
        if (fulls)
          pixel = srcp0[x] * factor; // 0..255->0..1
        else
          pixel = (srcp0[x] - half) * factor + 0.5f; // back to 0..1.0 (0.5 centered)
#else
        if (fulls)
          pixel = srcp0[x] * factor - 0.5f; // 0..1->-0.5..0.5
        else
          pixel = (srcp0[x] - half) * factor; // -0.5..0.5 when fulld
#endif
			}
			else {
        pixel = (srcp0[x] - limit_lo_s) * factor + limit_lo_d / 255.0f;
			}
			dstp0[x] = pixel;
		}
		dstp0 += dst_pitch;
		srcp0 += src_pitch;
	}
	// seems we better stuck with C in the future on such a simple loops
	// if we could put it in a separate file
	// VS2015 AVX2 code for this:
	// takes (8 uint16_t -> 8*float(256 bit) at a time) * unroll_by_2
	// then makes singles with unrolled_by_4 until it can, then do the rest.
	/*
	AVX2 by VS2015: (8*uint16->8*float)xUnrollBy2
	$LL7@convert_ui:
	vpmovzxwd ymm0, XMMWORD PTR [esi+ecx*2]
	vcvtdq2ps ymm0, ymm0
	vmulps	ymm0, ymm0, ymm2
	vmovups	YMMWORD PTR [edi+ecx*4], ymm0
	vpmovzxwd ymm0, XMMWORD PTR [esi+ecx*2+16]
	vcvtdq2ps ymm0, ymm0
	vmulps	ymm0, ymm0, ymm2
	vmovups	YMMWORD PTR [edi+ecx*4+32], ymm0
	add	ecx, 16					; 00000010H
	cmp	ecx, ebx
	jl	SHORT $LL7@convert_ui

	SSE2 by VS2015 (4*uint16->4*float)xUnrollBy2
	$LL7@convert_ui:
	movq	xmm1, QWORD PTR [ebp+ecx*2]
	xorps	xmm0, xmm0
	punpcklwd xmm1, xmm0
	cvtdq2ps xmm0, xmm1
	mulps	xmm0, xmm3
	movups	XMMWORD PTR [ebx+ecx*4], xmm0
	movq	xmm1, QWORD PTR [ebp+ecx*2+8]
	xorps	xmm0, xmm0
	punpcklwd xmm1, xmm0
	cvtdq2ps xmm0, xmm1
	mulps	xmm0, xmm3
	movups	XMMWORD PTR [ebx+ecx*4+16], xmm0
	add	ecx, 8
	cmp	ecx, esi
	jl	SHORT $LL7@convert_ui
	*/
}

BitDepthConvFuncPtr get_convert_to_8_function(bool full_scale, int source_bitdepth, int dither_mode, int dither_bitdepth, int rgb_step, int cpu)
{
	std::map<std::tuple<bool, int, int, int, int, int>, BitDepthConvFuncPtr> func_copy;
	using std::make_tuple;
	/*
	conv_function_full_scale = (sse2 && dither_mode<0) ? convert_rgb_uint16_to_8_c<10, -1, 8, 1> : (dither_mode>=0 ? convert_rgb_uint16_to_8_c<10, 0, 8, 1> : convert_rgb_uint16_to_8_c<10, -1, 8, 1>);
	conv_function_full_scale_no_dither = sse2 ? convert_rgb_uint16_to_8_c<10, -1, 8, 1> : convert_rgb_uint16_to_8_c<10, -1, 8, 1>;
	conv_function_shifted_scale = (sse2 && dither_mode<0) ? convert_uint16_to_8_sse2<10> : (dither_mode>=0 ? convert_uint16_to_8_c<10, 0, 8> : convert_uint16_to_8_c<10, -1, 8>);
	*/
	const int DITHER_TARGET_BITDEPTH_8 = 8;
	const int DITHER_TARGET_BITDEPTH_7 = 7;
	const int DITHER_TARGET_BITDEPTH_6 = 6;
	const int DITHER_TARGET_BITDEPTH_5 = 5;
	const int DITHER_TARGET_BITDEPTH_4 = 4;
	const int DITHER_TARGET_BITDEPTH_3 = 3;
	const int DITHER_TARGET_BITDEPTH_2 = 2;
	const int DITHER_TARGET_BITDEPTH_1 = 1;
	const int DITHER_TARGET_BITDEPTH_0 = 0;

	if (dither_mode < 0)
		dither_bitdepth = 8; // default entry in the tables below
	if (dither_mode == 1) // no special version for fullscale dithering down.
		full_scale = false;

	// full scale

	// no dither, C
	func_copy[make_tuple(true, 10, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<10, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 12, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<12, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 14, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<14, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<16, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	// for RGB48 and RGB64 source
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 3, 0)] = convert_rgb_uint16_to_8_c<16, -1, DITHER_TARGET_BITDEPTH_8, 1>; // dither rgb_step param is n/a
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 4, 0)] = convert_rgb_uint16_to_8_c<16, -1, DITHER_TARGET_BITDEPTH_8, 1>; // dither rgb_step param is n/a

																																																																				//-----------
																																																																				// full scale, no dither, SSE2
	func_copy[make_tuple(true, 10, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<10, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 12, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<12, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 14, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<14, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, -1, DITHER_TARGET_BITDEPTH_8, 1>;
	// for RGB48 and RGB64 source
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 3, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, -1, DITHER_TARGET_BITDEPTH_8, 1>; // dither rgb_step param is n/a
	func_copy[make_tuple(true, 16, -1, DITHER_TARGET_BITDEPTH_8, 4, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, -1, DITHER_TARGET_BITDEPTH_8, 1>; // dither rgb_step param is n/a

																																																																									 //-----------
																																																																									 // full scale, dither, C
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_6, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_rgb_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_4, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_rgb_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_2, 1>;

	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_6, 1>;
	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_rgb_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_4, 1>;

	func_copy[make_tuple(true, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<14, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_8_c<14, 0, DITHER_TARGET_BITDEPTH_6, 1>;

	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_8_c<16, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	// for RGB48 and RGB64 source
	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 3, 0)] = convert_rgb_uint16_to_8_c<16, 0, DITHER_TARGET_BITDEPTH_8, 3>; // dither rgb_step param is filled
	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 4, 0)] = convert_rgb_uint16_to_8_c<16, 0, DITHER_TARGET_BITDEPTH_8, 4>; // dither rgb_step param is filled

																																																																			//-----------
																																																																			// full scale, dither, SSE2
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<10, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<10, 0, DITHER_TARGET_BITDEPTH_6, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<10, 0, DITHER_TARGET_BITDEPTH_4, 1>;
	func_copy[make_tuple(true, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<10, 0, DITHER_TARGET_BITDEPTH_2, 1>;

	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<12, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<12, 0, DITHER_TARGET_BITDEPTH_6, 1>;
	func_copy[make_tuple(true, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<12, 0, DITHER_TARGET_BITDEPTH_4, 1>;

	func_copy[make_tuple(true, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<14, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	func_copy[make_tuple(true, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<14, 0, DITHER_TARGET_BITDEPTH_6, 1>;

	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, 0, DITHER_TARGET_BITDEPTH_8, 1>;
	// for RGB48 and RGB64 source
	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 3, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, 0, DITHER_TARGET_BITDEPTH_8, 3>; // dither rgb_step param is filled
	func_copy[make_tuple(true, 16, 0, DITHER_TARGET_BITDEPTH_8, 4, CPUF_SSE2)] = convert_rgb_uint16_to_8_sse2<16, 0, DITHER_TARGET_BITDEPTH_8, 4>; // dither rgb_step param is filled

																																																																								 //-----------
																																																																								 // Floyd dither, C, dither to 8 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_8>;
	// Floyd dither, C, dither to 7 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_7>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_7>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_7>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_7>;
	// Floyd dither, C, dither to 6 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_6>;
	// Floyd dither, C, dither to 5 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_5>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_5>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_5>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_5>;
	// Floyd dither, C, dither to 4 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_4>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_4>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_4>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_4>;
	// Floyd dither, C, dither to 3 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_3>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_3>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_3>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_3>;
	// Floyd dither, C, dither to 2 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_2>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_2>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_2>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_2>;
	// Floyd dither, C, dither to 1 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_1>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_1>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_1>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_1>;
	// Floyd dither, C, dither to 0 bits
	func_copy[make_tuple(false, 10, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 10, 8, DITHER_TARGET_BITDEPTH_0>;
	func_copy[make_tuple(false, 12, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 12, 8, DITHER_TARGET_BITDEPTH_0>;
	func_copy[make_tuple(false, 14, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 14, 8, DITHER_TARGET_BITDEPTH_0>;
	func_copy[make_tuple(false, 16, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint8_t, 16, 8, DITHER_TARGET_BITDEPTH_0>;

	// shifted scale (YUV)

	// no dither, C
	func_copy[make_tuple(false, 10, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<10, -1, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 12, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<12, -1, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 14, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<14, -1, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 16, -1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<16, -1, DITHER_TARGET_BITDEPTH_8>;
	// no dither, SSE2
	func_copy[make_tuple(false, 10, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_sse2<10>;
	func_copy[make_tuple(false, 12, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_sse2<12>;
	func_copy[make_tuple(false, 14, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_sse2<14>;
	func_copy[make_tuple(false, 16, -1, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_sse2<16>;

	// dither, C, dither to 8 bits
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<14, 0, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_8_c<16, 0, DITHER_TARGET_BITDEPTH_8>;
	// dither, SSE2
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<10, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<12, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<14, DITHER_TARGET_BITDEPTH_8>;
	func_copy[make_tuple(false, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<16, DITHER_TARGET_BITDEPTH_8>;

	// dither, C, dither to 6 bits, max diff 8, allowed from 10-14 bits
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_8_c<14, 0, DITHER_TARGET_BITDEPTH_6>;
	// dither, SSE2
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<10, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<12, DITHER_TARGET_BITDEPTH_6>;
	func_copy[make_tuple(false, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<14, DITHER_TARGET_BITDEPTH_6>;

	// dither, C, dither to 4 bits, max diff 8, allowed from 10-12 bits
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_4>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint16_to_8_c<12, 0, DITHER_TARGET_BITDEPTH_4>;
	// dither, SSE2
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<10, DITHER_TARGET_BITDEPTH_4>;
	func_copy[make_tuple(false, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<12, DITHER_TARGET_BITDEPTH_4>;

	// dither, C, dither to 2 bits, max diff 8, allowed from 10 bits
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint16_to_8_c<10, 0, DITHER_TARGET_BITDEPTH_2>;
	// dither, SSE2
	func_copy[make_tuple(false, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, CPUF_SSE2)] = convert_uint16_to_8_dither_sse2<10, DITHER_TARGET_BITDEPTH_2>;

	BitDepthConvFuncPtr result = func_copy[make_tuple(full_scale, source_bitdepth, dither_mode, dither_bitdepth, rgb_step, cpu)];
	if (result == nullptr)
		result = func_copy[make_tuple(full_scale, source_bitdepth, dither_mode, dither_bitdepth, rgb_step, 0)]; // fallback to C
	return result;
}

BitDepthConvFuncPtr get_convert_to_16_16_down_dither_function(bool full_scale, int source_bitdepth, int target_bitdepth, int dither_mode, int dither_bitdepth, int rgb_step, int cpu)
{
	std::map<std::tuple<bool, int /*src*/, int /*target*/, int /*dithermode*/, int /*ditherbits*/, int /*rgbstep*/, int /*cpu*/>, BitDepthConvFuncPtr> func_copy;
	using std::make_tuple;
	/*
	conv_function_full_scale = (sse2 && dither_mode<0) ? convert_rgb_uint16_to_8_c<10, -1, 8, 1> : (dither_mode>=0 ? convert_rgb_uint16_to_8_c<10, 0, 8, 1> : convert_rgb_uint16_to_8_c<10, -1, 8, 1>);
	conv_function_full_scale_no_dither = sse2 ? convert_rgb_uint16_to_8_c<10, -1, 8, 1> : convert_rgb_uint16_to_8_c<10, -1, 8, 1>;
	conv_function_shifted_scale = (sse2 && dither_mode<0) ? convert_uint16_to_8_sse2<10> : (dither_mode>=0 ? convert_uint16_to_8_c<10, 0, 8> : convert_uint16_to_8_c<10, -1, 8>);
	*/
	const int DITHER_TARGET_BITDEPTH_14 = 14;
	const int DITHER_TARGET_BITDEPTH_12 = 12;
	const int DITHER_TARGET_BITDEPTH_10 = 10;
	const int DITHER_TARGET_BITDEPTH_8 = 8;
	const int DITHER_TARGET_BITDEPTH_7 = 7;
	const int DITHER_TARGET_BITDEPTH_6 = 6;
	const int DITHER_TARGET_BITDEPTH_5 = 5;
	const int DITHER_TARGET_BITDEPTH_4 = 4;
	const int DITHER_TARGET_BITDEPTH_3 = 3;
	const int DITHER_TARGET_BITDEPTH_2 = 2; // only for 10->10 bits, but dithering_bits==2
	const int DITHER_TARGET_BITDEPTH_1 = 1; // FloydSteinberg allows any difference in the implementation
	const int DITHER_TARGET_BITDEPTH_0 = 0; // FloydSteinberg allows any difference in the implementation

	if (dither_mode == 1) // no special version for fullscale dithering down.
		full_scale = false;

	if (full_scale) {
		// 16->10,12,14
		// dither, C, dither to N bits
		func_copy[make_tuple(true, 16, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 16, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 10, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(true, 16, 12, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(true, 16, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 16, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(true, 16, 14, 0, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 14, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(true, 16, 14, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(true, 16, 14, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 16, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(true, 16, 16, 0, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(true, 16, 16, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(true, 16, 16, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 16, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_8>;

		// 14->10,12
		// dither, C, dither to N bits
		func_copy[make_tuple(true, 14, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 14, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 14, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_6>;

		func_copy[make_tuple(true, 14, 12, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(true, 14, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 14, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 14, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_6>;

		func_copy[make_tuple(true, 14, 14, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(true, 14, 14, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 14, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 14, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_6>;

		// 12->10
		// dither, C, dither to N bits
		func_copy[make_tuple(true, 12, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 12, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 12, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(true, 12, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_4>;

		func_copy[make_tuple(true, 12, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(true, 12, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 12, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(true, 12, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_4>;

		// 10->10
		// dither, C, dither to N bits
		func_copy[make_tuple(true, 10, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(true, 10, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(true, 10, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(true, 10, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_rgb_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_2>;
	}
	else {

		// floyd 16->
		// 16->10,12,14
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 16, 10, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 10, DITHER_TARGET_BITDEPTH_0>;

		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 16, 12, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 12, DITHER_TARGET_BITDEPTH_0>;

		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 16, 14, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 14, DITHER_TARGET_BITDEPTH_0>;
		// keeping bit depth but dither down
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 16, 16, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 16, 16, DITHER_TARGET_BITDEPTH_0>;
		// floyd 14->
		// 14->10,12
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 14, 10, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 10, DITHER_TARGET_BITDEPTH_0>;

		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 14, 12, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 12, DITHER_TARGET_BITDEPTH_0>;
		// keeping bit depth but dither down
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 14, 14, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 14, 14, DITHER_TARGET_BITDEPTH_0>;
		// floyd 12->
		// 12->10
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 12, 10, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 10, DITHER_TARGET_BITDEPTH_0>;
		// keeping bit depth but dither down
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 12, 12, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 12, 12, DITHER_TARGET_BITDEPTH_0>;
		// floyd 12->
		// 10->10
		// dither, C, dither to N bits
		// keeping bit depth but dither down
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_7, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_7>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_5, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_5>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_3, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_3>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_2>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_1, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_1>;
		func_copy[make_tuple(false, 10, 10, 1, DITHER_TARGET_BITDEPTH_0, 1, 0)] = convert_uint_floyd_c<uint16_t, uint16_t, 10, 10, DITHER_TARGET_BITDEPTH_0>;

		// end of floyd

		// shifted scale
		// 16->10,12,14
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 16, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 10, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(false, 16, 12, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(false, 16, 14, 0, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 14, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(false, 16, 14, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 14, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 12, DITHER_TARGET_BITDEPTH_8>;

		func_copy[make_tuple(false, 16, 16, 0, DITHER_TARGET_BITDEPTH_14, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_14>;
		func_copy[make_tuple(false, 16, 16, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 16, 16, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 16, 16, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<16, 16, DITHER_TARGET_BITDEPTH_8>;

		// 14->10,12
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 14, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 10, DITHER_TARGET_BITDEPTH_6>;

		func_copy[make_tuple(false, 14, 12, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 14, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 12, DITHER_TARGET_BITDEPTH_6>;

		func_copy[make_tuple(false, 14, 14, 0, DITHER_TARGET_BITDEPTH_12, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_12>;
		func_copy[make_tuple(false, 14, 14, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 14, 14, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 14, 14, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<14, 14, DITHER_TARGET_BITDEPTH_6>;

		// 12->10
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 12, 10, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 12, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 12, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 12, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 10, DITHER_TARGET_BITDEPTH_4>;

		func_copy[make_tuple(false, 12, 12, 0, DITHER_TARGET_BITDEPTH_10, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_10>;
		func_copy[make_tuple(false, 12, 12, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 12, 12, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 12, 12, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint16_to_uint16_dither_c<12, 12, DITHER_TARGET_BITDEPTH_4>;

		// 10->10 only dither down
		// dither, C, dither to N bits
		func_copy[make_tuple(false, 10, 10, 0, DITHER_TARGET_BITDEPTH_8, 1, 0)] = convert_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_8>;
		func_copy[make_tuple(false, 10, 10, 0, DITHER_TARGET_BITDEPTH_6, 1, 0)] = convert_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_6>;
		func_copy[make_tuple(false, 10, 10, 0, DITHER_TARGET_BITDEPTH_4, 1, 0)] = convert_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_4>;
		func_copy[make_tuple(false, 10, 10, 0, DITHER_TARGET_BITDEPTH_2, 1, 0)] = convert_uint16_to_uint16_dither_c<10, 10, DITHER_TARGET_BITDEPTH_2>;

	}
	BitDepthConvFuncPtr result = func_copy[make_tuple(full_scale, source_bitdepth, target_bitdepth, dither_mode, dither_bitdepth, rgb_step, cpu)];
	if (result == nullptr)
		result = func_copy[make_tuple(full_scale, source_bitdepth, target_bitdepth, dither_mode, dither_bitdepth, rgb_step, 0)]; // fallback to C
	return result;
}

template <int SRC_BITS, int TGT_BITS, bool DITHER, bool CHROMA>
BitDepthConvFuncPtr2 get_cuda_conv_bits()
{
	return ConvertBitsKernel<
		(SRC_BITS == TGT_BITS) ? 0 :
		(SRC_BITS == 32) ? 4 :
		(TGT_BITS == 32) ? 5 :
		(SRC_BITS < TGT_BITS) ? 3 :
		(DITHER) ? 1 : 2>::template launch<SRC_BITS, TGT_BITS, DITHER, CHROMA>;
}

template <int SRC_BITS, int TGT_BITS, bool DITHER>
BitDepthConvFuncPtr2 get_cuda_conv_bits(bool chroma)
{
	if (chroma) {
		return get_cuda_conv_bits<SRC_BITS, TGT_BITS, DITHER, true>();
	}
	return get_cuda_conv_bits<SRC_BITS, TGT_BITS, DITHER, false>();
}

template <int SRC_BITS, int TGT_BITS> 
BitDepthConvFuncPtr2 get_cuda_conv_bits(bool dither, bool chroma)
{
	if (dither) {
		return get_cuda_conv_bits<SRC_BITS, TGT_BITS, true>(chroma);
	}
	return get_cuda_conv_bits<SRC_BITS, TGT_BITS, false>(chroma);
}

template <int SRC_BITS>
BitDepthConvFuncPtr2 get_cuda_conv_bits(int tgt_bits, bool dither, bool chroma, PNeoEnv env)
{
	switch (tgt_bits) {
	case 8:
		return get_cuda_conv_bits<SRC_BITS, 8>(dither, chroma);
	case 10:
		return get_cuda_conv_bits<SRC_BITS, 10>(dither, chroma);
	case 12:
		return get_cuda_conv_bits<SRC_BITS, 12>(dither, chroma);
	case 14:
		return get_cuda_conv_bits<SRC_BITS, 14>(dither, chroma);
	case 16:
		return get_cuda_conv_bits<SRC_BITS, 16>(dither, chroma);
	case 32:
		return get_cuda_conv_bits<SRC_BITS, 32>(dither, chroma);
	}
	env->ThrowError("ConvertToBits: unsupported bit depth");
	return nullptr;
}

static BitDepthConvFuncPtr2 get_cuda_conv_bits(int pixelsize, int bits_per_pixel, int tgt_bits, bool dither, bool chroma, PNeoEnv env)
{
	if (pixelsize == 1) {
		return get_cuda_conv_bits<8>(tgt_bits, dither, chroma, env);
	}
	else if (pixelsize == 2) {
		switch (bits_per_pixel) {
		case 10:
			return get_cuda_conv_bits<10>(tgt_bits, dither, chroma, env);
		case 12:
			return get_cuda_conv_bits<12>(tgt_bits, dither, chroma, env);
		case 14:
			return get_cuda_conv_bits<14>(tgt_bits, dither, chroma, env);
		case 16:
			return get_cuda_conv_bits<16>(tgt_bits, dither, chroma, env);
		}
	}
	else if(pixelsize == 4) {
		return get_cuda_conv_bits<32>(tgt_bits, dither, chroma, env);
	}
	env->ThrowError("ConvertToBits: unsupported bit depth");
	return nullptr;
}

ConvertBits::ConvertBits(PClip _child, const int _dither_mode, const int _target_bitdepth, bool _truerange, bool _fulls, bool _fulld, int _dither_bitdepth, IScriptEnvironment* env) :
	GenericVideoFilter(_child), dither_mode(_dither_mode), target_bitdepth(_target_bitdepth), truerange(_truerange),
	fulls(_fulls), fulld(_fulld), dither_bitdepth(_dither_bitdepth)
{

	pixelsize = vi.ComponentSize();
	bits_per_pixel = vi.BitsPerComponent();
	format_change_only = false;

	bool sse2 = !!(env->GetCPUFlags() & CPUF_SSE2);
	bool sse4 = !!(env->GetCPUFlags() & CPUF_SSE4_1);
	bool avx = !!(env->GetCPUFlags() & CPUF_AVX);
	bool avx2 = !!(env->GetCPUFlags() & CPUF_AVX2);

	BitDepthConvFuncPtr conv_function_full_scale;
	BitDepthConvFuncPtr conv_function_full_scale_no_dither;
	BitDepthConvFuncPtr conv_function_shifted_scale;

	conv_function_chroma = nullptr; // used only for 32bit float

	// CUDA
	conv_cuda = get_cuda_conv_bits(pixelsize, bits_per_pixel, target_bitdepth, dither_mode >= 0, false, env);
	conv_cuda_chroma = get_cuda_conv_bits(pixelsize, bits_per_pixel, target_bitdepth, dither_mode >= 0, true, env);

  if (bits_per_pixel < 32 && target_bitdepth < 32) {
    // 32 bit source: fulls, fulld handled properly
    if (fulls != fulld)
      env->ThrowError("ConvertBits: fulls and fulld should be the same for non-32bit float formats");
  }

  // 8-16bit->32bits support fulls fulld, alpha is always full-full
#define convert_uintN_to_float_functions(uint_X_t, source_bits) \
      conv_function_a = convert_uintN_to_float_c<uint_X_t, source_bits, false, true, true>; /* full-full */ \
      if (fulls && fulld) { \
        conv_function = convert_uintN_to_float_c<uint_X_t, source_bits, false, true, true>; \
        conv_function_chroma = convert_uintN_to_float_c<uint_X_t, source_bits, true, true, true>; \
      } \
      else if (fulls && !fulld) { \
        conv_function = convert_uintN_to_float_c<uint_X_t, source_bits, false, true, false>; \
        conv_function_chroma = convert_uintN_to_float_c<uint_X_t, source_bits, true, true, false>; \
      } \
      else if (!fulls && fulld) { \
        conv_function = convert_uintN_to_float_c<uint_X_t, source_bits, false, false, true>; \
        conv_function_chroma = convert_uintN_to_float_c<uint_X_t, source_bits, true, false, true>; \
      } \
      else if (!fulls && !fulld) { \
        conv_function = convert_uintN_to_float_c<uint_X_t, source_bits, false, false, false>; \
        conv_function_chroma = convert_uintN_to_float_c<uint_X_t, source_bits, true, false, false>; \
      }

  // ConvertToFloat
  if (target_bitdepth == 32) {
    if (pixelsize == 1) // 8->32 bit
    {
      convert_uintN_to_float_functions(uint8_t, 8)
    }
    else if (pixelsize == 2) // 16->32 bit
    {
      if (vi.IsPlanar() && truerange)
      {
        switch (bits_per_pixel)
        {
        case 10: 
          convert_uintN_to_float_functions(uint16_t, 10);
          break;
        case 12: 
          convert_uintN_to_float_functions(uint16_t, 12);
          break;
        case 14: 
          convert_uintN_to_float_functions(uint16_t, 14);
          break;
        case 16: 
          convert_uintN_to_float_functions(uint16_t, 16);
          break;
        default: env->ThrowError("ConvertToFloat: unsupported bit depth");
        }
      }
      else {
        convert_uintN_to_float_functions(uint16_t, 16);
      }
    }
    else
      env->ThrowError("ConvertToFloat: internal error 32->32 is not valid here");

    conv_function_a = conv_function; 

		if (vi.NumComponents() == 1)
			vi.pixel_type = VideoInfo::CS_Y32;
		else if (vi.Is420())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA420PS : VideoInfo::CS_YUV420PS;
		else if (vi.Is422())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA422PS : VideoInfo::CS_YUV422PS;
		else if (vi.Is444())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA444PS : VideoInfo::CS_YUV444PS;
		else if (vi.IsPlanarRGB())
			vi.pixel_type = VideoInfo::CS_RGBPS;
		else if (vi.IsPlanarRGBA())
			vi.pixel_type = VideoInfo::CS_RGBAPS;
		else
			env->ThrowError("ConvertToFloat: unsupported color space");

		return;
	}
	// ConvertToFloat end

// 32bit->8-16bits support fulls fulld
#define convert_32_to_uintN_functions(uint_X_t, target_bits) \
      conv_function_a = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, false, true, true> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, false, true, true> : convert_32_to_uintN_c<uint_X_t, target_bits, false, true, true>; /* full-full */ \
      if (fulls && fulld) { \
        conv_function = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, false, true, true> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, false, true, true> : convert_32_to_uintN_c<uint_X_t, target_bits, false, true, true>; \
        conv_function_chroma = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, true, true, true> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, true, true, true> : convert_32_to_uintN_c<uint_X_t, target_bits, true, true, true>; \
      } \
      else if (fulls && !fulld) { \
        conv_function = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, false, true, false> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, false, true, false> : convert_32_to_uintN_c<uint_X_t, target_bits, false, true, false>; \
        conv_function_chroma = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, true, true, false> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, true, true, false> : convert_32_to_uintN_c<uint_X_t, target_bits, true, true, false>; \
      } \
      else if (!fulls && fulld) { \
        conv_function = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, false, false, true> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, false, false, true> : convert_32_to_uintN_c<uint_X_t, target_bits, false, false, true>; \
        conv_function_chroma = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, true, false, true> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, true, false, true> : convert_32_to_uintN_c<uint_X_t, target_bits, true, false, true>; \
      } \
      else if (!fulls && !fulld) { \
        conv_function = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, false, false, false> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, false, false, false> : convert_32_to_uintN_c<uint_X_t, target_bits, false, false, false>; \
        conv_function_chroma = avx2 ? convert_32_to_uintN_avx2<uint_X_t, target_bits, true, false, false> : sse4 ? convert_32_to_uintN_sse<uint_X_t, target_bits, true, false, false> : convert_32_to_uintN_c<uint_X_t, target_bits, true, false, false>; \
      }

	// ConvertTo16bit() (10, 12, 14, 16)
	// Conversion to uint16_t targets
	// planar YUV(A) and RGB(A):
	//   from 8 bit -> 10/12/14/16 with strict range expansion or expansion to 16
	//   from 10/12/14 -> 16 bit with strict source range (expansion from 10/12/14 to 16 bit) or just casting pixel_type
	//   from 16 bit -> 10/12/14 bit with strict target range (reducing range from 16 bit to 10/12/14 bits) or just casting pixel_type
	//   from float -> 10/12/14/16 with strict range expansion or expansion to 16
	// packed RGB:
	//   RGB24->RGB48, RGB32->RGB64
	if (target_bitdepth > 8 && target_bitdepth <= 16) {
		// 8,10-16,32 -> 16 bit
		if (pixelsize == 1) // 8->10-12-14-16 bit
		{
			if (truerange)
			{
				switch (target_bitdepth)
				{
				case 10:
					conv_function_full_scale = sse2 ? convert_rgb_8_to_uint16_sse2<10> : convert_rgb_8_to_uint16_c<10>;
					conv_function_shifted_scale = sse2 ? convert_8_to_uint16_sse2<10> : convert_8_to_uint16_c<10>;
					break;
				case 12:
					conv_function_full_scale = sse2 ? convert_rgb_8_to_uint16_sse2<12> : convert_rgb_8_to_uint16_c<12>;
					conv_function_shifted_scale = sse2 ? convert_8_to_uint16_sse2<12> : convert_8_to_uint16_c<12>;
					break;
				case 14:
					conv_function_full_scale = sse2 ? convert_rgb_8_to_uint16_sse2<14> : convert_rgb_8_to_uint16_c<14>;
					conv_function_shifted_scale = sse2 ? convert_8_to_uint16_sse2<14> : convert_8_to_uint16_c<14>;
					break;
				case 16:
					conv_function_full_scale = sse2 ? convert_rgb_8_to_uint16_sse2<16> : convert_rgb_8_to_uint16_c<16>;
					conv_function_shifted_scale = sse2 ? convert_8_to_uint16_sse2<16> : convert_8_to_uint16_c<16>;
					break;
				default: env->ThrowError("ConvertTo16bit: unsupported bit depth");
				}
			}
			else {
				conv_function_full_scale = sse2 ? convert_rgb_8_to_uint16_sse2<16> : convert_rgb_8_to_uint16_c<16>;
				conv_function_shifted_scale = sse2 ? convert_8_to_uint16_sse2<16> : convert_8_to_uint16_c<16>;
			}

			if (fulls)
				conv_function = conv_function_full_scale; // rgb default, RGB scaling is not shift by 2/4/6/8 as in YUV but like 0..255->0..65535
			else
				conv_function = conv_function_shifted_scale; // yuv default

			conv_function_a = conv_function_full_scale; // alpha copy is the same full scale
		}
		else if (pixelsize == 2)
		{
			// 10-16->10-16
			if (truerange)
			{

				// full_scale is used for alpha plane always (keep max opacity 255, 1023, 4095, 16383, 65535)

				// fill conv_function_full_scale and conv_function_shifted_scale
				// first get full_scale converter functions, normal and optional dithered
				if (bits_per_pixel >= target_bitdepth) // reduce range or dither down keeping bit-depth format
				{
					conv_function_full_scale = nullptr; // BitBlt in GetFrame

					if (bits_per_pixel == 16) { // 16->10/12/14 keep full range
						switch (target_bitdepth)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<16, 10, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<16, 10, false> : convert_rgb_uint16_to_uint16_c<16, 10>;
							break;
						case 12: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<16, 12, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<16, 12, false> : convert_rgb_uint16_to_uint16_c<16, 12>;
							break;
						case 14: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<16, 14, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<16, 14, false> : convert_rgb_uint16_to_uint16_c<16, 14>;
							break;
						}
					}
					else if (bits_per_pixel == 14) { // 14->10/12 keep full range
						switch (target_bitdepth)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<14, 10, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<14, 10, false> : convert_rgb_uint16_to_uint16_c<14, 10>;
							break;
						case 12: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<14, 12, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<14, 12, false> : convert_rgb_uint16_to_uint16_c<14, 12>;
							break;
						}
					}
					else if (bits_per_pixel == 12) { // 12->10 keep full range
						switch (target_bitdepth)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<12, 10, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<12, 10, false> : convert_rgb_uint16_to_uint16_c<12, 10>;
							break;
						}
					}

					conv_function_full_scale_no_dither = conv_function_full_scale; // save ditherless, used for possible alpha

					if (dither_mode >= 0) {
						conv_function_full_scale = get_convert_to_16_16_down_dither_function(true /*full scale*/, bits_per_pixel, target_bitdepth, dither_mode, dither_bitdepth, 1/*rgb_step n/a*/, 0 /*cpu none*/);
					}
				}
				else {// expand
							// no dither here
					if (target_bitdepth == 16) { // 10/12/14->16 keep full range
						switch (bits_per_pixel)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<10, 16, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<10, 16, false> : convert_rgb_uint16_to_uint16_c<10, 16>;
							break;
						case 12: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<12, 16, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<12, 16, false> : convert_rgb_uint16_to_uint16_c<12, 16>;
							break;
						case 14: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<14, 16, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<14, 16, false> : convert_rgb_uint16_to_uint16_c<14, 16>;
							break;
						}
					}
					else if (target_bitdepth == 14) { // 10/12->14 keep full range
						switch (bits_per_pixel)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<10, 14, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<10, 14, false> : convert_rgb_uint16_to_uint16_c<10, 14>;
							break;
						case 12: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<12, 14, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<12, 14, false> : convert_rgb_uint16_to_uint16_c<12, 14>;
							break;
						}
					}
					else if (target_bitdepth == 12) { // 10->12 keep full range
						switch (bits_per_pixel)
						{
						case 10: conv_function_full_scale = sse4 ? convert_rgb_uint16_to_uint16_sse2<10, 12, true> : sse2 ? convert_rgb_uint16_to_uint16_sse2<10, 12, false> : convert_rgb_uint16_to_uint16_c<10, 12>;
							break;
						}
					}

					conv_function_full_scale_no_dither = conv_function_full_scale; // save ditherless, used for possible alpha
				}

				// fill shift_range converter functions
				if (bits_per_pixel >= target_bitdepth) { // reduce range 16->14/12/10 14->12/10 12->10. template: bitshift
					if (dither_mode < 0) {
						switch (bits_per_pixel - target_bitdepth)
						{
						case 2:
							conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<false, 2> : avx ? convert_uint16_to_uint16_c_avx<false, 2> : (sse2 ? convert_uint16_to_uint16_sse2<false, 2> : convert_uint16_to_uint16_c<false, 2>);
							break;
						case 4:
							conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<false, 4> : avx ? convert_uint16_to_uint16_c_avx<false, 4> : (sse2 ? convert_uint16_to_uint16_sse2<false, 4> : convert_uint16_to_uint16_c<false, 4>);
							break;
						case 6:
							conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<false, 6> : avx ? convert_uint16_to_uint16_c_avx<false, 6> : (sse2 ? convert_uint16_to_uint16_sse2<false, 6> : convert_uint16_to_uint16_c<false, 6>);
							break;
						}
					}
					else {
						// dither
						conv_function_shifted_scale = get_convert_to_16_16_down_dither_function(false /*not full scale*/, bits_per_pixel, target_bitdepth, dither_mode, dither_bitdepth, 1/*rgb_step n/a*/, 0 /*cpu none*/);
					}
				}
				else { // expand range
					switch (target_bitdepth - bits_per_pixel)
					{
					case 2: conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<true, 2> : avx ? convert_uint16_to_uint16_c_avx<true, 2> : (sse2 ? convert_uint16_to_uint16_sse2<true, 2> : convert_uint16_to_uint16_c<true, 2>); break;
					case 4: conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<true, 4> : avx ? convert_uint16_to_uint16_c_avx<true, 4> : (sse2 ? convert_uint16_to_uint16_sse2<true, 4> : convert_uint16_to_uint16_c<true, 4>); break;
					case 6: conv_function_shifted_scale = avx2 ? convert_uint16_to_uint16_c_avx2<true, 6> : avx ? convert_uint16_to_uint16_c_avx<true, 6> : (sse2 ? convert_uint16_to_uint16_sse2<true, 6> : convert_uint16_to_uint16_c<true, 6>); break;
					}
				}
			}
			else {
				// no conversion for truerange == false
			}

			// 10/12/14 -> 16 bit or 16 bit -> 10/12/14 bit
			// range reducing or expansion (truerange=true), or just overriding the pixel_type, keeping scale at 16 bits
			// 10-16 -> 10->16 truerange == false already handled
			if (truerange) {
				if (fulls)
					conv_function = conv_function_full_scale; // rgb default, RGB scaling is not shift by 2/4/6/8 as in YUV but like 0..255->0..65535
				else
					conv_function = conv_function_shifted_scale; // yuv default

				conv_function_a = conv_function_full_scale_no_dither; // alpha copy is always full scale w/o dithering
			}
			else { // truerange==false
						 // 10->12 .. 16->12 etc
						 // only vi bit_depth format override
				format_change_only = true;
			}
		}
    else if (pixelsize == 4) // 32->10-16 bit
		{
			if (truerange) {
				switch (target_bitdepth)
				{
				case 10:
          convert_32_to_uintN_functions(uint16_t, 10); // all variations of fulls fulld
					break;
				case 12:
          convert_32_to_uintN_functions(uint16_t, 12);
					break;
				case 14:
          convert_32_to_uintN_functions(uint16_t, 14);
					break;
				case 16:
          convert_32_to_uintN_functions(uint16_t, 16);
					break;
				}
			}
			else {
        convert_32_to_uintN_functions(uint16_t, 16);
			}
		}
		else {
			env->ThrowError("ConvertTo16bit: unsupported bit depth");
		}

		// set output vi format
		if (vi.IsRGB24()) {
			if (target_bitdepth == 16)
				vi.pixel_type = VideoInfo::CS_BGR48;
			else
				env->ThrowError("ConvertTo16bit: unsupported bit depth");
		}
		else if (vi.IsRGB32()) {
			if (target_bitdepth == 16)
				vi.pixel_type = VideoInfo::CS_BGR64;
			else
				env->ThrowError("ConvertTo16bit: unsupported bit depth");
		}
		else {
			// Y or YUV(A) or PlanarRGB(A)
			if (vi.IsYV12()) // YV12 can have an exotic compatibility constant
				vi.pixel_type = VideoInfo::CS_YV12;
			int new_bitdepth_bits;
			switch (target_bitdepth) {
			case 8: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_8; break;
			case 10: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_10; break;
			case 12: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_12; break;
			case 14: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_14; break;
			case 16: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_16; break;
			case 32: new_bitdepth_bits = VideoInfo::CS_Sample_Bits_32; break;
			}
			vi.pixel_type = (vi.pixel_type & ~VideoInfo::CS_Sample_Bits_Mask) | new_bitdepth_bits;
		}

		return;
	}

	// ConvertTo8bit()
	if (target_bitdepth == 8) {
		if (pixelsize == 2) // 16(,14,12,10)->8 bit
		{
			// it gets complicated, so we better using tuples for function lookup
			// parameters for full scale: source bitdepth, dither_type (-1:none, 0:ordered), target_dither_bitdepth(default 8, 2,4,6), rgb_step(3 for RGB48, 4 for RGB64, 1 for all planars)
			// rgb_step can differ from 1 only when source bits_per_pixel==16 and packed RGB type
			// target_dither_bitdepth==8 (RFU for dithering down from e.g. 10->2 bit)

			if (dither_mode == 0 && (dither_bitdepth != 2 && dither_bitdepth != 4 && dither_bitdepth != 6 && dither_bitdepth != 8))
				env->ThrowError("ConvertBits: invalid dither target bitdepth %d", dither_bitdepth);

			// fill conv_function_full_scale and conv_function_shifted_scale
			// conv_function_full_scale_no_dither: for alpha plane
			if (truerange) {
				conv_function_full_scale = get_convert_to_8_function(true, bits_per_pixel, dither_mode, dither_bitdepth, 1, CPUF_SSE2);
				conv_function_full_scale_no_dither = get_convert_to_8_function(true, bits_per_pixel, -1, dither_bitdepth /* n/a */, 1, CPUF_SSE2); // force dither_mode==-1
				conv_function_shifted_scale = get_convert_to_8_function(false, bits_per_pixel, dither_mode, dither_bitdepth, 1, CPUF_SSE2);
			}
			else {
				conv_function_full_scale = get_convert_to_8_function(true, 16, dither_mode, dither_bitdepth, 1, CPUF_SSE2);
				conv_function_full_scale_no_dither = get_convert_to_8_function(true, 16, -1, dither_bitdepth /* n/a */, 1, CPUF_SSE2);
				conv_function_shifted_scale = get_convert_to_8_function(false, 16, dither_mode, dither_bitdepth, 1, CPUF_SSE2);
			}

			// override for RGB48 and 64 (internal rgb_step may differ when dithering is used
			if (vi.IsRGB48()) { // packed RGB: specify rgb_step 3 or 4 for dither table access
				conv_function_full_scale = get_convert_to_8_function(true, 16, dither_mode, dither_bitdepth, 3, CPUF_SSE2);
      } else if(vi.IsRGB64()) {
				conv_function_full_scale = get_convert_to_8_function(true, 16, dither_mode, dither_bitdepth, 4, CPUF_SSE2);
			}

			// packed RGB scaling is full_scale 0..65535->0..255
			if (fulls)
				conv_function = conv_function_full_scale; // rgb default, RGB scaling is not shift by 2/4/6/8 as in YUV but like 0..255->0..65535
			else
				conv_function = conv_function_shifted_scale; // yuv default

			conv_function_a = conv_function_full_scale_no_dither; // alpha copy is the same full scale, w/o dithering

		}
		else if (vi.ComponentSize() == 4) // 32->8 bit, no dithering option atm
		{
      convert_32_to_uintN_functions(uint8_t, 8); // all combinations of fulls, fulld
		}
		else
			env->ThrowError("ConvertTo8bit: unsupported bit depth");

		if (vi.NumComponents() == 1)
			vi.pixel_type = VideoInfo::CS_Y8;
		else if (vi.Is420())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA420 : VideoInfo::CS_YV12;
		else if (vi.Is422())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA422 : VideoInfo::CS_YV16;
		else if (vi.Is444())
			vi.pixel_type = vi.IsYUVA() ? VideoInfo::CS_YUVA444 : VideoInfo::CS_YV24;
		else if (vi.IsRGB48())
			vi.pixel_type = VideoInfo::CS_BGR24;
		else if (vi.IsRGB64())
			vi.pixel_type = VideoInfo::CS_BGR32;
		else if (vi.IsPlanarRGB())
			vi.pixel_type = VideoInfo::CS_RGBP;
		else if (vi.IsPlanarRGBA())
			vi.pixel_type = VideoInfo::CS_RGBAP;
		else
			env->ThrowError("ConvertTo8bit: unsupported color space");

		return;
	}

	env->ThrowError("ConvertBits: unsupported target bit-depth (%d)", target_bitdepth);

}

AVSValue __cdecl ConvertBits::Create(AVSValue args, void* user_data, IScriptEnvironment* env) {
	PClip clip = args[0].AsClip();
	//0   1        2        3         4         5           6    
	//c[bits]i[truerange]b[dither]i[dither_bits]i[fulls]b[fulld]b

	const VideoInfo &vi = clip->GetVideoInfo();

	int create_param = (int)reinterpret_cast<intptr_t>(user_data);

	// bits parameter is compulsory
	if (!args[1].Defined() && create_param == 0) {
		env->ThrowError("ConvertBits: missing bits parameter");
	}

	// when converting from/true 10-16 bit formats, truerange=false indicates bitdepth of 16 bits regardless of the 10-12-14 bit format
	bool assume_truerange = args[2].AsBool(true); // n/a for non planar formats
																								// bits parameter

	int target_bitdepth = args[1].AsInt(create_param); // default comes by calling from old To8,To16,ToFloat functions
	int source_bitdepth = vi.BitsPerComponent();
	int dither_bitdepth = args[4].AsInt(target_bitdepth);

	if (target_bitdepth != 8 && target_bitdepth != 10 && target_bitdepth != 12 && target_bitdepth != 14 && target_bitdepth != 16 && target_bitdepth != 32)
		env->ThrowError("ConvertBits: invalid bit depth: %d", target_bitdepth);

	if (create_param == 8 && target_bitdepth != 8)
		env->ThrowError("ConvertTo8Bit: invalid bit depth: %d", target_bitdepth);
	if (create_param == 32 && target_bitdepth != 32)
		env->ThrowError("ConvertToFloat: invalid bit depth: %d", target_bitdepth);
	if (create_param == 16 && (target_bitdepth == 8 || target_bitdepth == 32))
		env->ThrowError("ConvertTo16bit: invalid bit depth: %d", target_bitdepth);

	if (args[2].Defined()) {
		if (!vi.IsPlanar())
			env->ThrowError("ConvertBits: truerange specified for non-planar source");
	}

	// override defaults, e.g. set full range for greyscale clip conversion that is RGB
  // Post 2664: can be set. Full range is default also for float (and cannot be set to false)
  bool fulls = args[5].AsBool(vi.IsRGB()/* || ((target_bitdepth == 32 || source_bitdepth == 32))*/);
	bool fulld = args[6].AsBool(fulls);

	int dither_type = args[3].AsInt(-1);
	bool dither_defined = args[3].Defined();
	if (dither_defined && dither_type != 1 && dither_type != 0 && dither_type != -1)
    env->ThrowError("ConvertBits: invalid dither type parameter. Only -1 (disabled), 0 (ordered dither) or 1 (Floyd-S) is allowed");

	if (dither_type >= 0) {
		if (source_bitdepth < target_bitdepth)
			env->ThrowError("ConvertBits: dithering is allowed only for scale down");
		if (dither_bitdepth > target_bitdepth)
			env->ThrowError("ConvertBits: dither_bits must be <= target bitdepth");
		if (target_bitdepth == 32)
			env->ThrowError("ConvertBits: dithering is not allowed only for 32 bit targets");
	}

	if (dither_type == 0) {

		if (source_bitdepth == 32)
			env->ThrowError("ConvertBits: dithering is not allowed only for 32 bit sources");

		if (dither_bitdepth < 2 || dither_bitdepth > 16)
			env->ThrowError("ConvertBits: invalid dither_bits specified");

		if (dither_bitdepth % 2)
			env->ThrowError("ConvertBits: dither_bits must be even");

		if (source_bitdepth - dither_bitdepth > 8)
			env->ThrowError("ConvertBits: dither_bits cannot differ with more than 8 bits from source");

		if (source_bitdepth == 8)
			env->ThrowError("ConvertBits: dithering is not supported for 8 bit sources");
	}

	// floyd
	if (dither_type == 1) {

		if (source_bitdepth == 8 || source_bitdepth == 32)
			env->ThrowError("ConvertBits: Floyd-S: dithering is allowed only for 10-16 bit sources");

		if (dither_bitdepth < 0 || dither_bitdepth > 16)
			env->ThrowError("ConvertBits: Floyd-S: invalid dither_bits specified");

		if ((dither_bitdepth > 8 && (dither_bitdepth % 2) != 0)) // must be even above 8 bits. 0 is ok, means real b/w
			env->ThrowError("ConvertBits: Floyd-S: dither_bits must be 0..8, 10, 12, 14, 16");
	}

	// no change -> return unmodified if no dithering required, or dither bitdepth is the same as target
	if (source_bitdepth == target_bitdepth) { // 10->10 .. 16->16
		if (dither_type < 0 || dither_bitdepth == target_bitdepth)
			return clip;
		if (vi.IsRGB() && !vi.IsPlanar())
			env->ThrowError("ConvertBits: dithering_bits should be the as target bitdepth for packed RGB formats");
		// here: we allow e.g. a 16->16 bit conversion with dithering bitdepth of 8
	}

	// YUY2 conversion is limited
	if (vi.IsYUY2()) {
		env->ThrowError("ConvertBits: YUY2 source is 8-bit only");
	}

	if (vi.IsYV411()) {
		env->ThrowError("ConvertBits: YV411 source cannot be converted");
	}

	// packed RGB conversion is limited
	if (vi.IsRGB24() || vi.IsRGB32()) {
		if (target_bitdepth != 16)
			env->ThrowError("ConvertBits: invalid bit-depth specified for packed RGB");
	}

	if (vi.IsRGB48() || vi.IsRGB64()) {
		if (target_bitdepth != 8)
			env->ThrowError("ConvertBits: invalid bit-depth specified for packed RGB");
	}

	// remark
	// source_10_bit.ConvertTo16bit(truerange=true)  : upscale range
	// source_10_bit.ConvertTo16bit(truerange=false) : leaves data, only format conversion
	// source_10_bit.ConvertTo16bit(bits=12,truerange=true)  : upscale range from 10 to 12
	// source_10_bit.ConvertTo16bit(bits=12,truerange=false) : leaves data, only format conversion
	// source_16_bit.ConvertTo16bit(bits=10, truerange=true)  : downscale range
	// source_16_bit.ConvertTo16bit(bits=10, truerange=false) : leaves data, only format conversion

  if (fulls != fulld && target_bitdepth != 32 && source_bitdepth != 32)
    env->ThrowError("ConvertBits: fulls must be the same as fulld for non 32bit target and source");

	return new ConvertBits(clip, dither_type, target_bitdepth, assume_truerange, fulls, fulld, dither_bitdepth, env);
}


PVideoFrame __stdcall ConvertBits::GetFrame(int n, IScriptEnvironment* env_) 
{
	PNeoEnv env = env_;
	PVideoFrame src = child->GetFrame(n, env);

	if (format_change_only)
	{
		// for 10-16 bit: simple format override in constructor
		return src;
	}

	PVideoFrame dst = env->NewVideoFrame(vi);

	if (vi.IsPlanar())
	{
		int planes_y[4] = { PLANAR_Y, PLANAR_U, PLANAR_V, PLANAR_A };
		int planes_r[4] = { PLANAR_G, PLANAR_B, PLANAR_R, PLANAR_A };
		int *planes = (vi.IsYUV() || vi.IsYUVA()) ? planes_y : planes_r;
		for (int p = 0; p < vi.NumComponents(); ++p) {
			const int plane = planes[p];
			if (IS_CUDA) {
				const bool chroma = (plane == PLANAR_U || plane == PLANAR_V);
				(chroma ? conv_cuda_chroma : conv_cuda)(
					src->GetReadPtr(plane), dst->GetWritePtr(plane),
					src->GetRowSize(plane), src->GetHeight(plane),
					src->GetPitch(plane), dst->GetPitch(plane), env);
				DEBUG_SYNC;
			}
			else if (plane == PLANAR_A) {
				if (conv_function_a == nullptr)
					env->BitBlt(dst->GetWritePtr(plane), dst->GetPitch(plane), src->GetReadPtr(plane), src->GetPitch(plane), src->GetRowSize(plane), src->GetHeight(plane));
				else
					conv_function_a(src->GetReadPtr(plane), dst->GetWritePtr(plane),
						src->GetRowSize(plane), src->GetHeight(plane),
						src->GetPitch(plane), dst->GetPitch(plane));
			}
			else if (conv_function == nullptr)
				env->BitBlt(dst->GetWritePtr(plane), dst->GetPitch(plane), src->GetReadPtr(plane), src->GetPitch(plane), src->GetRowSize(plane), src->GetHeight(plane));
			else {
				const bool chroma = (plane == PLANAR_U || plane == PLANAR_V);
				if (chroma && conv_function_chroma != nullptr)
					// 32bit float needs separate conversion (possible chroma -0.5 .. 0.5 option)
					// until then the conv_function_ch behaves the same as conv_function
          // see #ifdef FLOAT_CHROMA_IS_HALF_CENTERED
					conv_function_chroma(src->GetReadPtr(plane), dst->GetWritePtr(plane),
						src->GetRowSize(plane), src->GetHeight(plane),
						src->GetPitch(plane), dst->GetPitch(plane));
				else
					conv_function(src->GetReadPtr(plane), dst->GetWritePtr(plane),
						src->GetRowSize(plane), src->GetHeight(plane),
						src->GetPitch(plane), dst->GetPitch(plane));
			}
		}
	}
	else {
		// packed RGBs
		if (IS_CUDA) {
			conv_cuda(src->GetReadPtr(), dst->GetWritePtr(),
				src->GetRowSize(), src->GetHeight(),
				src->GetPitch(), dst->GetPitch(), env);
			DEBUG_SYNC;
		}
		else {
			conv_function(src->GetReadPtr(), dst->GetWritePtr(),
				src->GetRowSize(), src->GetHeight(),
				src->GetPitch(), dst->GetPitch());
		}
	}
	return dst;
}

